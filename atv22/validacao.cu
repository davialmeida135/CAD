#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <omp.h>

// --- Configuration Constants ---
#define GRID_SIZE_X 512        // Number of grid points in X dimension
#define GRID_SIZE_Y 512        // Number of grid points in Y dimension
#define GRID_SIZE_Z 512        // Number of grid points in Z dimension
#define NUM_TIME_STEPS 10      // Total number of simulation time steps
#define CELL_SPACING_X 1.0f     // Spatial step size (delta X)
#define CELL_SPACING_Y 1.0f     // Spatial step size (delta Y)
#define CELL_SPACING_Z 1.0f     // Spatial step size (delta Z)
#define TIME_STEP_INCREMENT 0.1f // Time step size (delta T)
#define DIFFUSION_COEFFICIENT 0.5f // Diffusion coefficient (Nu)
#define PERTURBATION_MAGNITUDE 100.0f // Magnitude of perturbation
#define RADIUS 1
#define BLOCK_SIZE_X 8 // Number of threads in X dimension per block
#define BLOCK_SIZE_Y 8 // Number of threads in Y dimension per block
#define BLOCK_SIZE_Z 8 // Number of threads in Z dimension per block

// Macro to convert 3D grid coordinates to a 1D array index
#define MAP_3D_TO_1D_INDEX(x_coord, y_coord, z_coord) ((z_coord) * GRID_SIZE_Y * GRID_SIZE_X + (y_coord) * GRID_SIZE_X + (x_coord))

__global__ void update_temperature_field(double* temperature_field_new, double* temperature_field_old, 
    int grid_points_x, int grid_points_y, int grid_points_z, double diffusion_alpha) {

    // Calcula as coordenadas 3D da thread atual dentro do grid
    // Cada thread processará um ponto específico da grade 3D
    // Block Idx = posição do bloco no grid
    // Block Dim = número de threads por bloco
    // Thread Idx = posição da thread dentro do bloco
    int thread_x = blockIdx.x * blockDim.x + threadIdx.x;  // Posição X da thread no grid global
    int thread_y = blockIdx.y * blockDim.y + threadIdx.y;  // Posição Y da thread no grid global
    int thread_z = blockIdx.z * blockDim.z + threadIdx.z;  // Posição Z da thread no grid global

    // Verifica se a thread está dentro dos limites válidos da grade
    // Exclui as bordas (pontos nas extremidades) pois eles não têm todos os vizinhos necessários
    // Para aplicar o operador de diferenças finitas, precisamos de 6 vizinhos (±X, ±Y, ±Z)
    if (thread_x > 0 && thread_x < grid_points_x-1 &&     // Não está na borda esquerda/direita
        thread_y > 0 && thread_y < grid_points_y-1 &&     // Não está na borda frente/trás  
        thread_z > 0 && thread_z < grid_points_z-1) {     // Não está na borda superior/inferior
        
        // Converte as coordenadas 3D (x,y,z) para índice 1D no array
        // Fórmula: z*altura*largura + y*largura + x
        // Isso mapeia a grade 3D para um array linear na memória
        int current_index = thread_z * grid_points_y * grid_points_x + thread_y * grid_points_x + thread_x;
        
        // Calcula os índices dos 6 vizinhos próximos (norte, sul, leste, oeste, cima, baixo)
        int neighbor_x_minus = current_index - 1;
        int neighbor_x_plus = current_index + 1;
        int neighbor_y_minus = current_index - grid_points_x;
        int neighbor_y_plus = current_index + grid_points_x;
        int neighbor_z_minus = current_index - grid_points_x * grid_points_y;
        int neighbor_z_plus = current_index + grid_points_x * grid_points_y;
        
        // Aplica a equação
        temperature_field_new[current_index] = temperature_field_old[current_index] + diffusion_alpha * (
            temperature_field_old[neighbor_x_plus] + temperature_field_old[neighbor_x_minus] +   
            temperature_field_old[neighbor_y_plus] + temperature_field_old[neighbor_y_minus] +    
            temperature_field_old[neighbor_z_plus] + temperature_field_old[neighbor_z_minus] -   
            6 * temperature_field_old[current_index]);                                           
    }
}

float calculate_max_deviation_from_one(double* temperature_field, int grid_points_x, int grid_points_y, int grid_points_z) {
    float max_deviation = 0.0f;
    int total_points = grid_points_x * grid_points_y * grid_points_z;
    
    for (int point_index = 0; point_index < total_points; point_index++) {
        float current_deviation = fabs(temperature_field[point_index] - 1.0);
        if (current_deviation > max_deviation) {
            max_deviation = current_deviation;
        }
    }
    return max_deviation;
}

float run_cuda_simulation(int grid_points_x, int grid_points_y, int grid_points_z, int number_of_time_steps, double diffusion_alpha) {
    size_t memory_size = grid_points_x * grid_points_y * grid_points_z * sizeof(double);
    
    // Allocate host memory
    double *host_temperature_initial = (double*)malloc(memory_size);
    double *host_temperature_result = (double*)malloc(memory_size);
    
    if (!host_temperature_initial || !host_temperature_result) {
        printf("Failed to allocate host memory\n");
        return -1.0f;
    }
    
    // Initialize host array: all points to 1.0
    int total_grid_points = grid_points_x * grid_points_y * grid_points_z;
    for (int point_index = 0; point_index < total_grid_points; point_index++) {
        host_temperature_initial[point_index] = 1.0;
    }
    
    // Add perturbation at center
    int center_x_coordinate = grid_points_x / 2;
    int center_y_coordinate = grid_points_y / 2;
    int center_z_coordinate = grid_points_z / 2;
    int center_point_index = center_z_coordinate * grid_points_y * grid_points_x + 
                            center_y_coordinate * grid_points_x + center_x_coordinate;
    host_temperature_initial[center_point_index] += PERTURBATION_MAGNITUDE;
    
    // Allocate device memory
    double *device_temperature_old, *device_temperature_new;
    hipMalloc(&device_temperature_old, memory_size);
    hipMalloc(&device_temperature_new, memory_size);
    
    // Copy initial data to device
    hipMemcpy(device_temperature_old, host_temperature_initial, memory_size, hipMemcpyHostToDevice);
    
    // Define 3D block and grid
    dim3 threads_per_block(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    dim3 blocks_per_grid((grid_points_x + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, 
                        (grid_points_y + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, 
                        (grid_points_z + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
    
    printf("Starting CUDA simulation...\n");
    printf("Grid size: %dx%dx%d\n", grid_points_x, grid_points_y, grid_points_z);
    printf("Block size: %dx%dx%d\n", BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    printf("Grid dimensions: %dx%dx%d\n", blocks_per_grid.x, blocks_per_grid.y, blocks_per_grid.z);
    
    // Create CUDA events for timing
    hipEvent_t simulation_start_event, simulation_stop_event;
    hipEventCreate(&simulation_start_event);
    hipEventCreate(&simulation_stop_event);
    
    // Record start event
    hipEventRecord(simulation_start_event);
    
    // Main simulation loop
    for (int time_step = 0; time_step < number_of_time_steps; time_step++) {
        // Launch kernel
        update_temperature_field<<<blocks_per_grid, threads_per_block>>>(
            device_temperature_new, device_temperature_old, 
            grid_points_x, grid_points_y, grid_points_z, diffusion_alpha);
        
        // Check for kernel launch errors
        hipError_t kernel_error = hipGetLastError();
        if (kernel_error != hipSuccess) {
            printf("CUDA kernel error: %s\n", hipGetErrorString(kernel_error));
            break;
        }
        
        // Swap pointers
        double* temporary_pointer = device_temperature_old;
        device_temperature_old = device_temperature_new;
        device_temperature_new = temporary_pointer;
    }
    
    // Record stop event
    hipEventRecord(simulation_stop_event);
    
    // Wait for all kernels to complete
    hipEventSynchronize(simulation_stop_event);
    
    // Calculate elapsed time using CUDA events
    float elapsed_milliseconds = 0;
    hipEventElapsedTime(&elapsed_milliseconds, simulation_start_event, simulation_stop_event);
    float elapsed_seconds = elapsed_milliseconds / 1000.0f; // Convert to seconds
    
    // Copy result back to host
    hipMemcpy(host_temperature_result, device_temperature_old, memory_size, hipMemcpyDeviceToHost);
    
    // Calculate final deviation
    float final_maximum_deviation = calculate_max_deviation_from_one(host_temperature_result, grid_points_x, grid_points_y, grid_points_z);
    
    printf("Total CUDA simulation time: %f seconds\n", elapsed_seconds);
    printf("Final max deviation from 1.0: %.6f\n", final_maximum_deviation);
    printf("Center value: %.6f\n", host_temperature_result[center_point_index]);
    // Resultado CPU 7.388917

    // Cleanup CUDA events
    hipEventDestroy(simulation_start_event);
    hipEventDestroy(simulation_stop_event);
    
    // Cleanup memory
    hipFree(device_temperature_old);
    hipFree(device_temperature_new);
    free(host_temperature_initial);
    free(host_temperature_result);
    
    return elapsed_seconds;
}
// Adicione esta função CPU ao seu navier.cu
void update_temperature_cpu(double* temp_new, double* temp_old, 
                           int nx, int ny, int nz, double alpha) {
    #pragma omp parallel for num_threads(8)
    for (int z = 1; z < nz - 1; z++) {
        for (int y = 1; y < ny - 1; y++) {
            for (int x = 1; x < nx - 1; x++) {
                int idx = z * ny * nx + y * nx + x;
                int idx_xm = idx - 1;
                int idx_xp = idx + 1;
                int idx_ym = idx - nx;
                int idx_yp = idx + nx;
                int idx_zm = idx - nx * ny;
                int idx_zp = idx + nx * ny;
                
                temp_new[idx] = temp_old[idx] + alpha * (
                    temp_old[idx_xm] + temp_old[idx_xp] +
                    temp_old[idx_ym] + temp_old[idx_yp] +
                    temp_old[idx_zm] + temp_old[idx_zp] -
                    6.0 * temp_old[idx]);
            }
        }
    }
}

float run_cpu_simulation(int nx, int ny, int nz, int num_steps, double alpha) {
    size_t size = nx * ny * nz * sizeof(double);
    
    double *cpu_temp_curr = (double*)malloc(size);
    double *cpu_temp_next = (double*)malloc(size);
    
    if (!cpu_temp_curr || !cpu_temp_next) {
        printf("Failed to allocate CPU memory\n");
        return -1.0f;
    }
    
    // Initialize: all points to 1.0
    for (int i = 0; i < nx * ny * nz; i++) {
        cpu_temp_curr[i] = 1.0;
    }
    
    // Add perturbation at center
    int center_idx = (nz/2) * ny * nx + (ny/2) * nx + (nx/2);
    cpu_temp_curr[center_idx] += PERTURBATION_MAGNITUDE;
    
    // Start timing
    struct timeval start, end;
    gettimeofday(&start, NULL);
    
    // Simulation loop
    for (int t = 0; t < num_steps; t++) {
        update_temperature_cpu(cpu_temp_next, cpu_temp_curr, nx, ny, nz, alpha);
        
        // Swap pointers
        double* temp = cpu_temp_curr;
        cpu_temp_curr = cpu_temp_next;
        cpu_temp_next = temp;
    }
    
    // End timing
    gettimeofday(&end, NULL);
    double elapsed = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) * 1e-6;
    
    // Calculate results
    float max_dev = calculate_max_deviation_from_one(cpu_temp_curr, nx, ny, nz);
    
    printf("CPU Results:\n");
    printf("Total CPU simulation time: %f seconds\n", elapsed);
    printf("Final max deviation from 1.0: %.6f\n", max_dev);
    printf("Center value: %.6f\n", cpu_temp_curr[center_idx]);
    
    free(cpu_temp_curr);
    free(cpu_temp_next);
    
    return elapsed;
}
void compare_results(double* gpu_result, double* cpu_result, int nx, int ny, int nz, double tolerance = 1e-6) {
    int total_points = nx * ny * nz;
    int differences = 0;
    double max_error = 0.0;
    double sum_error = 0.0;
    
    for (int i = 0; i < total_points; i++) {
        double error = fabs(gpu_result[i] - cpu_result[i]);
        if (error > tolerance) {
            differences++;
        }
        if (error > max_error) {
            max_error = error;
        }
        sum_error += error;
    }
    
    double avg_error = sum_error / total_points;
    
    printf("\n=== VALIDATION RESULTS ===\n");
    printf("Total points: %d\n", total_points);
    printf("Points with differences > %.2e: %d (%.2f%%)\n", 
           tolerance, differences, 100.0 * differences / total_points);
    printf("Maximum error: %.2e\n", max_error);
    printf("Average error: %.2e\n", avg_error);
}


// Adicione estas funções antes do main()

// Estrutura para armazenar resultados da simulação
typedef struct {
    double* temperature_field;
    float execution_time;
    float max_deviation;
    double center_value;
} SimulationResult;

// Função para executar simulação CPU e retornar resultados
SimulationResult run_cpu_simulation_with_results(int nx, int ny, int nz, int num_steps, double alpha) {
    SimulationResult result = {0};
    size_t size = nx * ny * nz * sizeof(double);
    
    result.temperature_field = (double*)malloc(size);
    double *cpu_temp_next = (double*)malloc(size);
    
    if (!result.temperature_field || !cpu_temp_next) {
        printf("Failed to allocate CPU memory\n");
        result.execution_time = -1.0f;
        return result;
    }
    
    // Initialize: all points to 1.0
    for (int i = 0; i < nx * ny * nz; i++) {
        result.temperature_field[i] = 1.0;
    }
    
    // Add perturbation at center
    int center_idx = (nz/2) * ny * nx + (ny/2) * nx + (nx/2);
    result.temperature_field[center_idx] += PERTURBATION_MAGNITUDE;
    
    // Start timing
    struct timeval start, end;
    gettimeofday(&start, NULL);
    
    // Simulation loop
    for (int t = 0; t < num_steps; t++) {
        update_temperature_cpu(cpu_temp_next, result.temperature_field, nx, ny, nz, alpha);
        
        // Swap pointers
        double* temp = result.temperature_field;
        result.temperature_field = cpu_temp_next;
        cpu_temp_next = temp;
    }
    
    // End timing
    gettimeofday(&end, NULL);
    result.execution_time = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) * 1e-6;
    
    // Calculate results
    result.max_deviation = calculate_max_deviation_from_one(result.temperature_field, nx, ny, nz);
    result.center_value = result.temperature_field[center_idx];
    
    printf("CPU Results:\n");
    printf("Total CPU simulation time: %f seconds\n", result.execution_time);
    printf("Final max deviation from 1.0: %.6f\n", result.max_deviation);
    printf("Center value: %.6f\n", result.center_value);
    
    free(cpu_temp_next);
    return result;
}

// Função para executar simulação GPU e retornar resultados
SimulationResult run_cuda_simulation_with_results(int grid_points_x, int grid_points_y, int grid_points_z, int number_of_time_steps, double diffusion_alpha) {
    SimulationResult result = {0};
    size_t memory_size = grid_points_x * grid_points_y * grid_points_z * sizeof(double);
    
    // Allocate host memory
    double *host_temperature_initial = (double*)malloc(memory_size);
    result.temperature_field = (double*)malloc(memory_size);
    
    if (!host_temperature_initial || !result.temperature_field) {
        printf("Failed to allocate host memory\n");
        result.execution_time = -1.0f;
        return result;
    }
    
    // Initialize host array: all points to 1.0
    int total_grid_points = grid_points_x * grid_points_y * grid_points_z;
    for (int point_index = 0; point_index < total_grid_points; point_index++) {
        host_temperature_initial[point_index] = 1.0;
    }
    
    // Add perturbation at center
    int center_x_coordinate = grid_points_x / 2;
    int center_y_coordinate = grid_points_y / 2;
    int center_z_coordinate = grid_points_z / 2;
    int center_point_index = center_z_coordinate * grid_points_y * grid_points_x + 
                            center_y_coordinate * grid_points_x + center_x_coordinate;
    host_temperature_initial[center_point_index] += PERTURBATION_MAGNITUDE;
    
    // Allocate device memory
    double *device_temperature_old, *device_temperature_new;
    hipMalloc(&device_temperature_old, memory_size);
    hipMalloc(&device_temperature_new, memory_size);
    
    // Copy initial data to device
    hipMemcpy(device_temperature_old, host_temperature_initial, memory_size, hipMemcpyHostToDevice);
    
    // Define 3D block and grid
    dim3 threads_per_block(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    dim3 blocks_per_grid((grid_points_x + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, 
                        (grid_points_y + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, 
                        (grid_points_z + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
    
    printf("Starting CUDA simulation...\n");
    printf("Grid size: %dx%dx%d\n", grid_points_x, grid_points_y, grid_points_z);
    printf("Block size: %dx%dx%d\n", BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    printf("Grid dimensions: %dx%dx%d\n", blocks_per_grid.x, blocks_per_grid.y, blocks_per_grid.z);
    
    // Create CUDA events for timing
    hipEvent_t simulation_start_event, simulation_stop_event;
    hipEventCreate(&simulation_start_event);
    hipEventCreate(&simulation_stop_event);
    
    // Record start event
    hipEventRecord(simulation_start_event);
    
    // Main simulation loop
    for (int time_step = 0; time_step < number_of_time_steps; time_step++) {
        // Launch kernel
        update_temperature_field<<<blocks_per_grid, threads_per_block>>>(
            device_temperature_new, device_temperature_old, 
            grid_points_x, grid_points_y, grid_points_z, diffusion_alpha);
        
        // Check for kernel launch errors
        hipError_t kernel_error = hipGetLastError();
        if (kernel_error != hipSuccess) {
            printf("CUDA kernel error: %s\n", hipGetErrorString(kernel_error));
            break;
        }
        
        // Swap pointers
        double* temporary_pointer = device_temperature_old;
        device_temperature_old = device_temperature_new;
        device_temperature_new = temporary_pointer;
    }
    
    // Record stop event
    hipEventRecord(simulation_stop_event);
    
    // Wait for all kernels to complete
    hipEventSynchronize(simulation_stop_event);
    
    // Calculate elapsed time using CUDA events
    float elapsed_milliseconds = 0;
    hipEventElapsedTime(&elapsed_milliseconds, simulation_start_event, simulation_stop_event);
    result.execution_time = elapsed_milliseconds / 1000.0f; // Convert to seconds
    
    // Copy result back to host
    hipMemcpy(result.temperature_field, device_temperature_old, memory_size, hipMemcpyDeviceToHost);
    
    // Calculate final deviation
    result.max_deviation = calculate_max_deviation_from_one(result.temperature_field, grid_points_x, grid_points_y, grid_points_z);
    result.center_value = result.temperature_field[center_point_index];
    
    printf("Total CUDA simulation time: %f seconds\n", result.execution_time);
    printf("Final max deviation from 1.0: %.6f\n", result.max_deviation);
    printf("Center value: %.6f\n", result.center_value);
    
    // Cleanup CUDA events
    hipEventDestroy(simulation_start_event);
    hipEventDestroy(simulation_stop_event);
    
    // Cleanup memory
    hipFree(device_temperature_old);
    hipFree(device_temperature_new);
    free(host_temperature_initial);
    
    return result;
}

// Função completa de validação com múltiplas métricas
typedef struct {
    bool validation_passed;
    int total_points;
    int points_with_differences;
    double max_absolute_error;
    double mean_absolute_error;
    double root_mean_square_error;
    double relative_error_percent;
    double correlation_coefficient;
    double tolerance_used;
} ValidationMetrics;

ValidationMetrics validate_simulation_results(SimulationResult cpu_result, SimulationResult gpu_result, 
                                             int nx, int ny, int nz, double tolerance = 1e-6) {
    ValidationMetrics metrics = {0};
    int total_points = nx * ny * nz;
    metrics.total_points = total_points;
    metrics.tolerance_used = tolerance;
    
    double sum_absolute_error = 0.0;
    double sum_squared_error = 0.0;
    double sum_cpu = 0.0;
    double sum_gpu = 0.0;
    double sum_cpu_squared = 0.0;
    double sum_gpu_squared = 0.0;
    double sum_cpu_gpu = 0.0;
    
    // Calculate all error metrics in one pass
    for (int i = 0; i < total_points; i++) {
        double cpu_val = cpu_result.temperature_field[i];
        double gpu_val = gpu_result.temperature_field[i];
        
        double absolute_error = fabs(gpu_val - cpu_val);
        
        // Count points exceeding tolerance
        if (absolute_error > tolerance) {
            metrics.points_with_differences++;
        }
        
        // Track maximum error
        if (absolute_error > metrics.max_absolute_error) {
            metrics.max_absolute_error = absolute_error;
        }
        
        // Accumulate for various metrics
        sum_absolute_error += absolute_error;
        sum_squared_error += absolute_error * absolute_error;
        
        // For correlation coefficient
        sum_cpu += cpu_val;
        sum_gpu += gpu_val;
        sum_cpu_squared += cpu_val * cpu_val;
        sum_gpu_squared += gpu_val * gpu_val;
        sum_cpu_gpu += cpu_val * gpu_val;
    }
    
    // Calculate derived metrics
    metrics.mean_absolute_error = sum_absolute_error / total_points;
    metrics.root_mean_square_error = sqrt(sum_squared_error / total_points);
    
    // Relative error as percentage
    double mean_cpu = sum_cpu / total_points;
    if (mean_cpu != 0.0) {
        metrics.relative_error_percent = (metrics.mean_absolute_error / fabs(mean_cpu)) * 100.0;
    }
    
    // Correlation coefficient (Pearson)
    double mean_cpu_calc = sum_cpu / total_points;
    double mean_gpu_calc = sum_gpu / total_points;
    
    double numerator = sum_cpu_gpu - total_points * mean_cpu_calc * mean_gpu_calc;
    double denominator = sqrt((sum_cpu_squared - total_points * mean_cpu_calc * mean_cpu_calc) * 
                             (sum_gpu_squared - total_points * mean_gpu_calc * mean_gpu_calc));
    
    if (denominator != 0.0) {
        metrics.correlation_coefficient = numerator / denominator;
    } else {
        metrics.correlation_coefficient = 1.0; // Perfect correlation if both are constant
    }
    
    // Determine if validation passed
    metrics.validation_passed = (metrics.max_absolute_error < tolerance) && 
                               (metrics.correlation_coefficient > 0.99999); // Very high correlation required
    
    return metrics;
}

void print_validation_report(ValidationMetrics metrics, SimulationResult cpu_result, SimulationResult gpu_result) {
    printf("\n=== VALIDATION REPORT ===\n");
    printf("========================================\n");
    
    printf("Dataset Statistics:\n");
    printf("  Total points analyzed: %d\n", metrics.total_points);
    printf("  Tolerance threshold: %.2e\n", metrics.tolerance_used);
    printf("\n");
    
    printf("Error Analysis:\n");
    printf("  Maximum absolute error: %.2e\n", metrics.max_absolute_error);
    printf("  Mean absolute error: %.2e\n", metrics.mean_absolute_error);
    printf("  Root mean square error: %.2e\n", metrics.root_mean_square_error);
    printf("  Relative error: %.6f%%\n", metrics.relative_error_percent);
    printf("  Points exceeding tolerance: %d (%.4f%%)\n", 
           metrics.points_with_differences, 
           100.0 * metrics.points_with_differences / metrics.total_points);
    printf("\n");
    
    printf("Statistical Correlation:\n");
    printf("  Pearson correlation coefficient: %.10f\n", metrics.correlation_coefficient);
    printf("\n");
    
    printf("Key Value Comparison:\n");
    printf("  CPU center value: %.10f\n", cpu_result.center_value);
    printf("  GPU center value: %.10f\n", gpu_result.center_value);
    printf("  Center value difference: %.2e\n", fabs(cpu_result.center_value - gpu_result.center_value));
    printf("  CPU max deviation: %.10f\n", cpu_result.max_deviation);
    printf("  GPU max deviation: %.10f\n", gpu_result.max_deviation);
    printf("  Max deviation difference: %.2e\n", fabs(cpu_result.max_deviation - gpu_result.max_deviation));
    printf("\n");
    
    printf("Performance Comparison:\n");
    printf("  CPU execution time: %.6f seconds\n", cpu_result.execution_time);
    printf("  GPU execution time: %.6f seconds\n", gpu_result.execution_time);
    if (cpu_result.execution_time > 0 && gpu_result.execution_time > 0) {
        printf("  Speedup (CPU/GPU): %.2fx\n", cpu_result.execution_time / gpu_result.execution_time);
    }
    printf("\n");
    
    printf("VALIDATION RESULT: %s\n", metrics.validation_passed ? "✓ PASSED" : "✗ FAILED");
    
    if (!metrics.validation_passed) {
        printf("\nPossible causes of validation failure:\n");
        if (metrics.max_absolute_error >= metrics.tolerance_used) {
            printf("  - Numerical errors exceed tolerance\n");
        }
        if (metrics.correlation_coefficient < 0.99999) {
            printf("  - Low correlation between CPU and GPU results\n");
        }
        printf("  - Consider adjusting tolerance or checking implementation\n");
    }
    
    printf("========================================\n");
}

// Função para salvar resultados detalhados (opcional)
void save_detailed_comparison(SimulationResult cpu_result, SimulationResult gpu_result, 
                             ValidationMetrics metrics, int nx, int ny, int nz) {
    FILE *file = fopen("validation_report.txt", "w");
    if (!file) {
        printf("Warning: Could not create detailed report file\n");
        return;
    }
    
    fprintf(file, "CUDA Validation Report\n");
    fprintf(file, "======================\n\n");
    fprintf(file, "Grid dimensions: %dx%dx%d\n", nx, ny, nz);
    fprintf(file, "Total points: %d\n", metrics.total_points);
    
    fprintf(file, "Error Metrics:\n");
    fprintf(file, "Max absolute error: %.2e\n", metrics.max_absolute_error);
    fprintf(file, "Mean absolute error: %.2e\n", metrics.mean_absolute_error);
    fprintf(file, "RMSE: %.2e\n", metrics.root_mean_square_error);
    fprintf(file, "Correlation: %.10f\n", metrics.correlation_coefficient);
    
    fprintf(file, "Performance:\n");
    fprintf(file, "CPU time: %.6f s\n", cpu_result.execution_time);
    fprintf(file, "GPU time: %.6f s\n", gpu_result.execution_time);
    if (cpu_result.execution_time > 0 && gpu_result.execution_time > 0) {
        fprintf(file, "Speedup: %.2fx\n", cpu_result.execution_time / gpu_result.execution_time);
    }
    
    fclose(file);
    printf("Detailed report saved to: validation_report.txt\n");
}

// Modificar o main() para usar as novas funções
int main() {
    // Simulation parameters
    int nx = GRID_SIZE_X;
    int ny = GRID_SIZE_Y;
    int nz = GRID_SIZE_Z;
    int nt = NUM_TIME_STEPS;
    double alpha = DIFFUSION_COEFFICIENT * TIME_STEP_INCREMENT / 
                   (CELL_SPACING_X * CELL_SPACING_X);
    
    printf("## Validação CPU vs GPU - Difusão 3D ##\n");
    printf("Grid: %dx%dx%d\n", nx, ny, nz);
    printf("Time steps: %d\n", nt);
    printf("Alpha: %f\n", alpha);
    printf("========================================\n");
    
    // Check CUDA device
    int device_count;
    hipGetDeviceCount(&device_count);
    if (device_count == 0) {
        printf("No CUDA devices found! Running CPU only.\n");
        SimulationResult cpu_only = run_cpu_simulation_with_results(nx, ny, nz, nt, alpha);
        free(cpu_only.temperature_field);
        return 1;
    }
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Using GPU: %s\n", prop.name);
    printf("========================================\n");
    
    // Run both simulations
    printf("\n>>> Running CPU simulation...\n");
    SimulationResult cpu_result = run_cpu_simulation_with_results(nx, ny, nz, nt, alpha);
    
    printf("\n>>> Running GPU simulation...\n");
    SimulationResult gpu_result = run_cuda_simulation_with_results(nx, ny, nz, nt, alpha);
    
    // Validate results
    if (cpu_result.execution_time > 0 && gpu_result.execution_time > 0) {
        ValidationMetrics validation = validate_simulation_results(cpu_result, gpu_result, nx, ny, nz, 1e-6);
        print_validation_report(validation, cpu_result, gpu_result);
        save_detailed_comparison(cpu_result, gpu_result, validation, nx, ny, nz);
    }
    
    // Cleanup
    free(cpu_result.temperature_field);
    free(gpu_result.temperature_field);
    
    return 0;
}