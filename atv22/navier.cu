#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

// --- Configuration Constants ---
#define GRID_SIZE_X 512        // Number of grid points in X dimension
#define GRID_SIZE_Y 512        // Number of grid points in Y dimension
#define GRID_SIZE_Z 512        // Number of grid points in Z dimension
#define NUM_TIME_STEPS 5      // Total number of simulation time steps
#define CELL_SPACING_X 1.0f     // Spatial step size (delta X)
#define CELL_SPACING_Y 1.0f     // Spatial step size (delta Y)
#define CELL_SPACING_Z 1.0f     // Spatial step size (delta Z)
#define TIME_STEP_INCREMENT 0.1f // Time step size (delta T)
#define DIFFUSION_COEFFICIENT 0.1f // Diffusion coefficient (Nu)
#define PERTURBATION_MAGNITUDE 10.0f // Magnitude of perturbation

// Macro to convert 3D grid coordinates to a 1D array index
#define MAP_3D_TO_1D_INDEX(x_coord, y_coord, z_coord) ((z_coord) * GRID_SIZE_Y * GRID_SIZE_X + (y_coord) * GRID_SIZE_X + (x_coord))

__global__ void update_temperature_field(double* temperature_field_new, double* temperature_field_old, 
    int grid_points_x, int grid_points_y, int grid_points_z, double diffusion_alpha) {

    // Calcula as coordenadas 3D da thread atual dentro do grid
    // Cada thread processará um ponto específico da grade 3D
    // Block Idx = posição do bloco no grid
    // Block Dim = número de threads por bloco
    // Thread Idx = posição da thread dentro do bloco
    int thread_x = blockIdx.x * blockDim.x + threadIdx.x;  // Posição X da thread no grid global
    int thread_y = blockIdx.y * blockDim.y + threadIdx.y;  // Posição Y da thread no grid global
    int thread_z = blockIdx.z * blockDim.z + threadIdx.z;  // Posição Z da thread no grid global

    // Verifica se a thread está dentro dos limites válidos da grade
    // Exclui as bordas (pontos nas extremidades) pois eles não têm todos os vizinhos necessários
    // Para aplicar o operador de diferenças finitas, precisamos de 6 vizinhos (±X, ±Y, ±Z)
    if (thread_x > 0 && thread_x < grid_points_x-1 &&     // Não está na borda esquerda/direita
        thread_y > 0 && thread_y < grid_points_y-1 &&     // Não está na borda frente/trás  
        thread_z > 0 && thread_z < grid_points_z-1) {     // Não está na borda superior/inferior
        
        // Converte as coordenadas 3D (x,y,z) para índice 1D no array
        // Fórmula: z*altura*largura + y*largura + x
        // Isso mapeia a grade 3D para um array linear na memória
        int current_index = thread_z * grid_points_y * grid_points_x + thread_y * grid_points_x + thread_x;
        
        // Calcula os índices dos 6 vizinhos próximos (norte, sul, leste, oeste, cima, baixo)
        int neighbor_x_minus = current_index - 1;
        int neighbor_x_plus = current_index + 1;
        int neighbor_y_minus = current_index - grid_points_x;
        int neighbor_y_plus = current_index + grid_points_x;
        int neighbor_z_minus = current_index - grid_points_x * grid_points_y;
        int neighbor_z_plus = current_index + grid_points_x * grid_points_y;
        
        // Aplica a equação
        temperature_field_new[current_index] = temperature_field_old[current_index] + diffusion_alpha * (
            temperature_field_old[neighbor_x_plus] + temperature_field_old[neighbor_x_minus] +   
            temperature_field_old[neighbor_y_plus] + temperature_field_old[neighbor_y_minus] +    
            temperature_field_old[neighbor_z_plus] + temperature_field_old[neighbor_z_minus] -   
            6 * temperature_field_old[current_index]);                                           
    }
}

float calculate_max_deviation_from_one(double* temperature_field, int grid_points_x, int grid_points_y, int grid_points_z) {
    float max_deviation = 0.0f;
    int total_points = grid_points_x * grid_points_y * grid_points_z;
    
    for (int point_index = 0; point_index < total_points; point_index++) {
        float current_deviation = fabs(temperature_field[point_index] - 1.0);
        if (current_deviation > max_deviation) {
            max_deviation = current_deviation;
        }
    }
    return max_deviation;
}

float run_cuda_simulation(int grid_points_x, int grid_points_y, int grid_points_z, int number_of_time_steps, double diffusion_alpha) {
    size_t memory_size = grid_points_x * grid_points_y * grid_points_z * sizeof(double);
    
    // Allocate host memory
    double *host_temperature_initial = (double*)malloc(memory_size);
    double *host_temperature_result = (double*)malloc(memory_size);
    
    if (!host_temperature_initial || !host_temperature_result) {
        printf("Failed to allocate host memory\n");
        return -1.0f;
    }
    
    // Initialize host array: all points to 1.0
    int total_grid_points = grid_points_x * grid_points_y * grid_points_z;
    for (int point_index = 0; point_index < total_grid_points; point_index++) {
        host_temperature_initial[point_index] = 1.0;
    }
    
    // Add perturbation at center
    int center_x_coordinate = grid_points_x / 2;
    int center_y_coordinate = grid_points_y / 2;
    int center_z_coordinate = grid_points_z / 2;
    int center_point_index = center_z_coordinate * grid_points_y * grid_points_x + 
                            center_y_coordinate * grid_points_x + center_x_coordinate;
    host_temperature_initial[center_point_index] += PERTURBATION_MAGNITUDE;
    
    // Allocate device memory
    double *device_temperature_old, *device_temperature_new;
    hipMalloc(&device_temperature_old, memory_size);
    hipMalloc(&device_temperature_new, memory_size);

    // Copy initial data to device
    // Destino, origem, tamanho, direção da cópia
    hipMemcpy(device_temperature_old, host_temperature_initial, memory_size, hipMemcpyHostToDevice);
    
    // Define 3D block and grid
    int block_size_x = 8, block_size_y = 8, block_size_z = 8;
    dim3 threads_per_block(block_size_x, block_size_y, block_size_z);
    dim3 blocks_per_grid((grid_points_x + block_size_x - 1) / block_size_x, 
                        (grid_points_y + block_size_y - 1) / block_size_y, 
                        (grid_points_z + block_size_z - 1) / block_size_z);
    
    printf("Starting CUDA simulation...\n");
    printf("Grid size: %dx%dx%d\n", grid_points_x, grid_points_y, grid_points_z);
    printf("Block size: %dx%dx%d\n", block_size_x, block_size_y, block_size_z);
    printf("Grid dimensions: %dx%dx%d\n", blocks_per_grid.x, blocks_per_grid.y, blocks_per_grid.z);
    
    // Create CUDA events for timing
    hipEvent_t simulation_start_event, simulation_stop_event;
    hipEventCreate(&simulation_start_event);
    hipEventCreate(&simulation_stop_event);
    
    // Record start event
    hipEventRecord(simulation_start_event);
    
    // Main simulation loop
    for (int time_step = 0; time_step < number_of_time_steps; time_step++) {
        // Launch kernel
        update_temperature_field<<<blocks_per_grid, threads_per_block>>>(
            device_temperature_new, device_temperature_old, 
            grid_points_x, grid_points_y, grid_points_z, diffusion_alpha);
        
        // Swap pointers
        double* temporary_pointer = device_temperature_old;
        device_temperature_old = device_temperature_new;
        device_temperature_new = temporary_pointer;
        
        // Check for kernel launch errors
        hipError_t kernel_error = hipGetLastError();
        if (kernel_error != hipSuccess) {
            printf("CUDA kernel error: %s\n", hipGetErrorString(kernel_error));
            break;
        }
            
    }
    
    // Record stop event
    hipEventRecord(simulation_stop_event);
    
    // Wait for all kernels to complete
    hipEventSynchronize(simulation_stop_event);
    
    // Calculate elapsed time using CUDA events
    float elapsed_milliseconds = 0;
    hipEventElapsedTime(&elapsed_milliseconds, simulation_start_event, simulation_stop_event);
    float elapsed_seconds = elapsed_milliseconds / 1000.0f; // Convert to seconds
    
    // Copy result back to host
    hipMemcpy(host_temperature_result, device_temperature_old, memory_size, hipMemcpyDeviceToHost);
    
    // Calculate final deviation
    float final_maximum_deviation = calculate_max_deviation_from_one(host_temperature_result, grid_points_x, grid_points_y, grid_points_z);
    
    printf("Total CUDA simulation time: %f seconds\n", elapsed_seconds);
    printf("Final max deviation from 1.0: %.6f\n", final_maximum_deviation);
    printf("Center value: %.6f\n", host_temperature_result[center_point_index]);
    
    // Cleanup CUDA events
    hipEventDestroy(simulation_start_event);
    hipEventDestroy(simulation_stop_event);
    
    // Cleanup memory
    hipFree(device_temperature_old);
    hipFree(device_temperature_new);
    free(host_temperature_initial);
    free(host_temperature_result);
    
    return elapsed_seconds;
}

int main() {
    // Simulation parameters
    int grid_points_x = GRID_SIZE_X;
    int grid_points_y = GRID_SIZE_Y;
    int grid_points_z = GRID_SIZE_Z;
    int number_of_time_steps = NUM_TIME_STEPS;
    double diffusion_alpha = DIFFUSION_COEFFICIENT * TIME_STEP_INCREMENT / 
                           (CELL_SPACING_X * CELL_SPACING_X);
    
    printf("## Simulação de Difusão 3D com CUDA ##\n");
    printf("Grid: %dx%dx%d\n", grid_points_x, grid_points_y, grid_points_z);
    printf("Time steps: %d\n", number_of_time_steps);
    printf("Alpha: %f\n", diffusion_alpha);
    printf("----------------------------------------\n");
    
    // Check CUDA device
    int cuda_device_count;
    hipGetDeviceCount(&cuda_device_count);
    if (cuda_device_count == 0) {
        printf("No CUDA devices found!\n");
        return 1;
    }
    
    hipDeviceProp_t device_properties;
    hipGetDeviceProperties(&device_properties, 0);
    printf("Using device: %s\n", device_properties.name);
    printf("----------------------------------------\n");
    
    // Run simulation
    float execution_time = run_cuda_simulation(grid_points_x, grid_points_y, grid_points_z, number_of_time_steps, diffusion_alpha);
    
    if (execution_time > 0) {
        printf("----------------------------------------\n");
        printf("Simulation completed successfully!\n");
    } else {
        printf("Simulation failed!\n");
        return 1;
    }
    
    return 0;
}