#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

// --- Configuration Constants ---
#define GRID_SIZE_X 256        // Number of grid points in X dimension
#define GRID_SIZE_Y 256     // Number of grid points in Y dimension
#define GRID_SIZE_Z 256       // Number of grid points in Z dimension
#define NUM_TIME_STEPS 5      // Total number of simulation time steps
#define CELL_SPACING_X 1.0f     // Spatial step size (delta X)
#define CELL_SPACING_Y 1.0f     // Spatial step size (delta Y)
#define CELL_SPACING_Z 1.0f     // Spatial step size (delta Z)
#define TIME_STEP_INCREMENT 0.1f // Time step size (delta T)
#define DIFFUSION_COEFFICIENT 0.1f // Diffusion coefficient (Nu)
#define PERTURBATION_MAGNITUDE 10.0f // Magnitude of perturbation

// Macro to convert 3D grid coordinates to a 1D array index
#define MAP_3D_TO_1D_INDEX(x, y, z) ((z) * GRID_SIZE_Y * GRID_SIZE_X + (y) * GRID_SIZE_X + (x))

__global__ void atualiza(double* vnew, double* vold, 
    int nx, int ny, int nz, double alpha) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (x > 0 && x < nx-1 && y > 0 && y < ny-1 && z > 0 && z < nz-1) {
        int idx = z * ny * nx + y * nx + x;
        int xm = idx - 1;
        int xp = idx + 1;
        int ym = idx - nx;
        int yp = idx + nx;
        int zm = idx - nx * ny;
        int zp = idx + nx * ny;
        
        vnew[idx] = vold[idx] + alpha * (
            vold[xp] + vold[xm] +
            vold[yp] + vold[ym] +
            vold[zp] + vold[zm] - 6 * vold[idx]);
    }
}

float calculate_max_deviation_from_one(double* field, int nx, int ny, int nz) {
    float max_dev = 0.0f;
    for (int i = 0; i < nx * ny * nz; i++) {
        float dev = fabs(field[i] - 1.0);
        if (dev > max_dev) max_dev = dev;
    }
    return max_dev;
}

float loop_cuda(int nx, int ny, int nz, int num_steps, double alpha) {
    size_t size = nx * ny * nz * sizeof(double);
    
    // Allocate host memory
    double *h_vold = (double*)malloc(size);
    double *h_result = (double*)malloc(size);
    
    if (!h_vold || !h_result) {
        printf("Failed to allocate host memory\n");
        return -1.0f;
    }
    
    // Initialize host array: all points to 1.0
    for (int i = 0; i < nx * ny * nz; i++) {
        h_vold[i] = 1.0;
    }
    
    // Add perturbation at center
    int center_x = nx / 2;
    int center_y = ny / 2;
    int center_z = nz / 2;
    int center_idx = center_z * ny * nx + center_y * nx + center_x;
    h_vold[center_idx] += PERTURBATION_MAGNITUDE;
    
    // Allocate device memory
    double *d_vold, *d_vnew;
    hipMalloc(&d_vold, size);
    hipMalloc(&d_vnew, size);
    
    // Copy initial data to device
    hipMemcpy(d_vold, h_vold, size, hipMemcpyHostToDevice);
    
    // Define 3D block and grid
    int bx = 8, by = 8, bz = 8;
    dim3 threads(bx, by, bz);
    dim3 grid((nx + bx - 1) / bx, (ny + by - 1) / by, (nz + bz - 1) / bz);
    
    printf("Starting CUDA simulation...\n");
    printf("Grid size: %dx%dx%d\n", nx, ny, nz);
    printf("Block size: %dx%dx%d\n", bx, by, bz);
    printf("Grid dimensions: %dx%dx%d\n", grid.x, grid.y, grid.z);
    
    // Create CUDA events for timing
    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    
    // Record start event
    hipEventRecord(start_event);
    
    // Main simulation loop
    for (int t = 0; t < num_steps; t++) {
        // Launch kernel
        atualiza<<<grid, threads>>>(d_vnew, d_vold, nx, ny, nz, alpha);
        
        // Check for kernel launch errors
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA kernel error: %s\n", hipGetErrorString(err));
            break;
        }
        
        // Swap pointers
        double* tmp = d_vold;
        d_vold = d_vnew;
        d_vnew = tmp;
    }
    
    // Record stop event
    hipEventRecord(stop_event);
    
    // Wait for all kernels to complete
    hipEventSynchronize(stop_event);
    
    // Calculate elapsed time using CUDA events
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    float elapsed = milliseconds / 1000.0f; // Convert to seconds
    
    // Copy result back to host
    hipMemcpy(h_result, d_vold, size, hipMemcpyDeviceToHost);
    
    // Calculate final deviation
    float final_max_dev = calculate_max_deviation_from_one(h_result, nx, ny, nz);
    
    printf("Total CUDA simulation time: %f seconds\n", elapsed);
    printf("Final max deviation from 1.0: %.6f\n", final_max_dev);
    printf("Center value: %.6f\n", h_result[center_idx]);
    
    // Cleanup CUDA events
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    
    // Cleanup memory
    hipFree(d_vold);
    hipFree(d_vnew);
    free(h_vold);
    free(h_result);
    
    return elapsed;
}

int main() {
    // Simulation parameters
    int nx = GRID_SIZE_X;
    int ny = GRID_SIZE_Y;
    int nz = GRID_SIZE_Z;
    int nt = NUM_TIME_STEPS;
    double alpha = DIFFUSION_COEFFICIENT * TIME_STEP_INCREMENT / 
                   (CELL_SPACING_X * CELL_SPACING_X);
    
    printf("## Simulação de Difusão 3D com CUDA ##\n");
    printf("Grid: %dx%dx%d\n", nx, ny, nz);
    printf("Time steps: %d\n", nt);
    printf("Alpha: %f\n", alpha);
    printf("----------------------------------------\n");
    
    // Check CUDA device
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        printf("No CUDA devices found!\n");
        return 1;
    }
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Using device: %s\n", prop.name);
    printf("----------------------------------------\n");
    
    // Run simulation
    float exec_time = loop_cuda(nx, ny, nz, nt, alpha);
    
    if (exec_time > 0) {
        printf("----------------------------------------\n");
        printf("Simulation completed successfully!\n");
    } else {
        printf("Simulation failed!\n");
        return 1;
    }
    
    return 0;
}