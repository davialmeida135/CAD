#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

// --- Configuration Constants ---
#define GRID_SIZE_X 128        // Number of grid points in X dimension
#define GRID_SIZE_Y 128        // Number of grid points in Y dimension
#define GRID_SIZE_Z 128        // Number of grid points in Z dimension
#define NUM_TIME_STEPS 5      // Total number of simulation time steps
#define CELL_SPACING_X 1.0f     // Spatial step size (delta X)
#define CELL_SPACING_Y 1.0f     // Spatial step size (delta Y)
#define CELL_SPACING_Z 1.0f     // Spatial step size (delta Z)
#define TIME_STEP_INCREMENT 0.1f // Time step size (delta T)
#define DIFFUSION_COEFFICIENT 0.1f // Diffusion coefficient (Nu)
#define PERTURBATION_MAGNITUDE 10.0f // Magnitude of perturbation
#define RADIUS 1
#define BLOCK_SIZE_X 8 // Number of threads in X dimension per block
#define BLOCK_SIZE_Y 8 // Number of threads in Y dimension per block
#define BLOCK_SIZE_Z 8 // Number of threads in Z dimension per block

// Macro to convert 3D grid coordinates to a 1D array index
#define MAP_3D_TO_1D_INDEX(x_coord, y_coord, z_coord) ((z_coord) * GRID_SIZE_Y * GRID_SIZE_X + (y_coord) * GRID_SIZE_X + (x_coord))

__global__ void update_temperature_field(double* temperature_field_new, double* temperature_field_old, 
    int grid_points_x, int grid_points_y, int grid_points_z, double diffusion_alpha) {

    __shared__ float tile[BLOCK_SIZE_X+3*RADIUS][BLOCK_SIZE_Y+3*RADIUS][BLOCK_SIZE_Z+3*RADIUS]; // Shared memory tile for block
    // Calcula as coordenadas 3D da thread atual dentro do grid
    // Cada thread processará um ponto específico da grade 3D
    // Block Idx = posição do bloco no grid
    // Block Dim = número de threads por bloco
    // Thread Idx = posição da thread dentro do bloco
    int thread_x = blockIdx.x * BLOCK_SIZE_X + threadIdx.x;  // Posição X da thread no grid global
    int thread_y = blockIdx.y * BLOCK_SIZE_Y + threadIdx.y;  // Posição Y da thread no grid global
    int thread_z = blockIdx.z * BLOCK_SIZE_Z + threadIdx.z;  // Posição Z da thread no grid global

    int tile_x = threadIdx.x + RADIUS; // Posição X da thread dentro do bloco
    int tile_y = threadIdx.y + RADIUS; // Posição Y da thread dentro do bloco
    int tile_z = threadIdx.z + RADIUS; // Posição Z da thread dentro do bloco

    // Verifica se a thread está dentro dos limites válidos da grade
    // Exclui as bordas (pontos nas extremidades) pois eles não têm todos os vizinhos necessários
    // Para aplicar o operador de diferenças finitas, precisamos de 6 vizinhos (±X, ±Y, ±Z)
    if (thread_x > 0 && thread_x < grid_points_x-1 &&     // Não está na borda esquerda/direita
        thread_y > 0 && thread_y < grid_points_y-1 &&     // Não está na borda frente/trás  
        thread_z > 0 && thread_z < grid_points_z-1) {     // Não está na borda superior/inferior
        
        
        int current_index = thread_z * grid_points_y * grid_points_x + thread_y * grid_points_x + thread_x;
        
        // Copia pixel central
        if (tile_x < BLOCK_SIZE_X && tile_y < BLOCK_SIZE_Y && tile_z < BLOCK_SIZE_Z)
            tile[tile_z][tile_y][tile_x] = temperature_field_old[current_index];

        // Copia vizinhos para a memória compartilhada
        if (tile_x - 1 >= 0 && tile_x - 1 < BLOCK_SIZE_X && tile_y < BLOCK_SIZE_Y && tile_z < BLOCK_SIZE_Z)
            tile[tile_z][tile_y][tile_x - 1] = temperature_field_old[current_index - 1]; // Vizinho à esquerda
        if (tile_x + 1 < BLOCK_SIZE_X && tile_y < BLOCK_SIZE_Y && tile_z < BLOCK_SIZE_Z)
            tile[tile_z][tile_y][tile_x + 1] = temperature_field_old[current_index + 1]; // Vizinho à direita
        if (tile_y - 1 >= 0 && tile_x < BLOCK_SIZE_X && tile_z < BLOCK_SIZE_Z)
            tile[tile_z][tile_y - 1][tile_x] = temperature_field_old[current_index - grid_points_x]; // Vizinho acima
        if (tile_y + 1 < BLOCK_SIZE_Y && tile_x < BLOCK_SIZE_X && tile_z < BLOCK_SIZE_Z)
            tile[tile_z][tile_y + 1][tile_x] = temperature_field_old[current_index + grid_points_x]; // Vizinho abaixo
        if (tile_z - 1 >= 0 && tile_x < BLOCK_SIZE_X && tile_y < BLOCK_SIZE_Y)
            tile[tile_z - 1][tile_y][tile_x] = temperature_field_old[current_index - grid_points_x * grid_points_y]; // Vizinho atrás
        if (tile_z + 1 < BLOCK_SIZE_Z && tile_x < BLOCK_SIZE_X && tile_y < BLOCK_SIZE_Y)
            tile[tile_z + 1][tile_y][tile_x] = temperature_field_old[current_index + grid_points_x * grid_points_y]; // Vizinho à frente

        __syncthreads();
        
        // Aplica a equação no tile shared
        temperature_field_new[current_index] = tile[tile_z][tile_y][tile_x] + diffusion_alpha * (
            tile[tile_z][tile_y][tile_x - 1] + tile[tile_z][tile_y][tile_x + 1] +   
            tile[tile_z][tile_y - 1][tile_x] + tile[tile_z][tile_y + 1][tile_x] +    
            tile[tile_z - 1][tile_y][tile_x] + tile[tile_z + 1][tile_y][tile_x] -   
            6 * tile[tile_z][tile_y][tile_x]);                                         
    }
}

float calculate_max_deviation_from_one(double* temperature_field, int grid_points_x, int grid_points_y, int grid_points_z) {
    float max_deviation = 0.0f;
    int total_points = grid_points_x * grid_points_y * grid_points_z;
    
    for (int point_index = 0; point_index < total_points; point_index++) {
        float current_deviation = fabs(temperature_field[point_index] - 1.0);
        if (current_deviation > max_deviation) {
            max_deviation = current_deviation;
        }
    }
    return max_deviation;
}

float run_cuda_simulation(int grid_points_x, int grid_points_y, int grid_points_z, int number_of_time_steps, double diffusion_alpha) {
    size_t memory_size = grid_points_x * grid_points_y * grid_points_z * sizeof(double);
    
    // Allocate host memory
    double *host_temperature_initial = (double*)malloc(memory_size);
    double *host_temperature_result = (double*)malloc(memory_size);
    
    if (!host_temperature_initial || !host_temperature_result) {
        printf("Failed to allocate host memory\n");
        return -1.0f;
    }
    
    // Initialize host array: all points to 1.0
    int total_grid_points = grid_points_x * grid_points_y * grid_points_z;
    for (int point_index = 0; point_index < total_grid_points; point_index++) {
        host_temperature_initial[point_index] = 1.0;
    }
    
    // Add perturbation at center
    int center_x_coordinate = grid_points_x / 2;
    int center_y_coordinate = grid_points_y / 2;
    int center_z_coordinate = grid_points_z / 2;
    int center_point_index = center_z_coordinate * grid_points_y * grid_points_x + 
                            center_y_coordinate * grid_points_x + center_x_coordinate;
    host_temperature_initial[center_point_index] += PERTURBATION_MAGNITUDE;
    
    // Allocate device memory
    double *device_temperature_old, *device_temperature_new;
    hipMalloc(&device_temperature_old, memory_size);
    hipMalloc(&device_temperature_new, memory_size);
    
    // Copy initial data to device
    hipMemcpy(device_temperature_old, host_temperature_initial, memory_size, hipMemcpyHostToDevice);
    
    // Define 3D block and grid
    dim3 threads_per_block(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    dim3 blocks_per_grid((grid_points_x + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, 
                        (grid_points_y + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, 
                        (grid_points_z + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
    
    printf("Starting CUDA simulation...\n");
    printf("Grid size: %dx%dx%d\n", grid_points_x, grid_points_y, grid_points_z);
    printf("Block size: %dx%dx%d\n", BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    printf("Grid dimensions: %dx%dx%d\n", blocks_per_grid.x, blocks_per_grid.y, blocks_per_grid.z);
    
    // Create CUDA events for timing
    hipEvent_t simulation_start_event, simulation_stop_event;
    hipEventCreate(&simulation_start_event);
    hipEventCreate(&simulation_stop_event);
    
    // Record start event
    hipEventRecord(simulation_start_event);
    
    // Main simulation loop
    for (int time_step = 0; time_step < number_of_time_steps; time_step++) {
        // Launch kernel
        update_temperature_field<<<blocks_per_grid, threads_per_block>>>(
            device_temperature_new, device_temperature_old, 
            grid_points_x, grid_points_y, grid_points_z, diffusion_alpha);
        
        // Check for kernel launch errors
        hipError_t kernel_error = hipGetLastError();
        if (kernel_error != hipSuccess) {
            printf("CUDA kernel error: %s\n", hipGetErrorString(kernel_error));
            break;
        }
        
        // Swap pointers
        double* temporary_pointer = device_temperature_old;
        device_temperature_old = device_temperature_new;
        device_temperature_new = temporary_pointer;
    }
    
    // Record stop event
    hipEventRecord(simulation_stop_event);
    
    // Wait for all kernels to complete
    hipEventSynchronize(simulation_stop_event);
    
    // Calculate elapsed time using CUDA events
    float elapsed_milliseconds = 0;
    hipEventElapsedTime(&elapsed_milliseconds, simulation_start_event, simulation_stop_event);
    float elapsed_seconds = elapsed_milliseconds / 1000.0f; // Convert to seconds
    
    // Copy result back to host
    hipMemcpy(host_temperature_result, device_temperature_old, memory_size, hipMemcpyDeviceToHost);
    
    // Calculate final deviation
    float final_maximum_deviation = calculate_max_deviation_from_one(host_temperature_result, grid_points_x, grid_points_y, grid_points_z);
    
    printf("Total CUDA simulation time: %f seconds\n", elapsed_seconds);
    printf("Final max deviation from 1.0: %.6f\n", final_maximum_deviation);
    printf("Center value: %.6f\n", host_temperature_result[center_point_index]);
    // Resultado CPU 7.388917

    // Cleanup CUDA events
    hipEventDestroy(simulation_start_event);
    hipEventDestroy(simulation_stop_event);
    
    // Cleanup memory
    hipFree(device_temperature_old);
    hipFree(device_temperature_new);
    free(host_temperature_initial);
    free(host_temperature_result);
    
    return elapsed_seconds;
}

int main() {
    // Simulation parameters
    int grid_points_x = GRID_SIZE_X;
    int grid_points_y = GRID_SIZE_Y;
    int grid_points_z = GRID_SIZE_Z;
    int number_of_time_steps = NUM_TIME_STEPS;
    double diffusion_alpha = DIFFUSION_COEFFICIENT * TIME_STEP_INCREMENT / 
                           (CELL_SPACING_X * CELL_SPACING_X);
    
    printf("## Simulação de Difusão 3D com CUDA ##\n");
    printf("Grid: %dx%dx%d\n", grid_points_x, grid_points_y, grid_points_z);
    printf("Time steps: %d\n", number_of_time_steps);
    printf("Alpha: %f\n", diffusion_alpha);
    printf("----------------------------------------\n");
    
    // Check CUDA device
    int cuda_device_count;
    hipGetDeviceCount(&cuda_device_count);
    if (cuda_device_count == 0) {
        printf("No CUDA devices found!\n");
        return 1;
    }
    
    hipDeviceProp_t device_properties;
    hipGetDeviceProperties(&device_properties, 0);
    printf("Using device: %s\n", device_properties.name);
    printf("----------------------------------------\n");
    
    // Run simulation
    float execution_time = run_cuda_simulation(grid_points_x, grid_points_y, grid_points_z, number_of_time_steps, diffusion_alpha);
    
    if (execution_time > 0) {
        printf("----------------------------------------\n");
        printf("Simulation completed successfully!\n");
    } else {
        printf("Simulation failed!\n");
        return 1;
    }
    
    return 0;
}