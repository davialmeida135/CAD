#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

// --- Configuration Constants ---
#define GRID_SIZE_X 128        // Number of grid points in X dimension
#define GRID_SIZE_Y 128        // Number of grid points in Y dimension
#define GRID_SIZE_Z 128        // Number of grid points in Z dimension
#define NUM_TIME_STEPS 5      // Total number of simulation time steps
#define CELL_SPACING_X 1.0f     // Spatial step size (delta X)
#define CELL_SPACING_Y 1.0f     // Spatial step size (delta Y)
#define CELL_SPACING_Z 1.0f     // Spatial step size (delta Z)
#define TIME_STEP_INCREMENT 0.1f // Time step size (delta T)
#define DIFFUSION_COEFFICIENT 0.1f // Diffusion coefficient (Nu)
#define PERTURBATION_MAGNITUDE 10.0f // Magnitude of perturbation
#define RADIUS 1
#define BLOCK_SIZE_X 8 // Number of threads in X dimension per block
#define BLOCK_SIZE_Y 8 // Number of threads in Y dimension per block
#define BLOCK_SIZE_Z 8 // Number of threads in Z dimension per block

// Macro to convert 3D grid coordinates to a 1D array index
#define MAP_3D_TO_1D_INDEX(x_coord, y_coord, z_coord) ((z_coord) * GRID_SIZE_Y * GRID_SIZE_X + (y_coord) * GRID_SIZE_X + (x_coord))

__global__ void update_temperature_field(double* temperature_field_new, double* temperature_field_old, 
    int grid_points_x, int grid_points_y, int grid_points_z, double diffusion_alpha) {

    __shared__ float tile[BLOCK_SIZE_X+3*RADIUS][BLOCK_SIZE_Y+3*RADIUS][BLOCK_SIZE_Z+3*RADIUS]; // Shared memory tile for block
    // Calcula as coordenadas 3D da thread atual dentro do grid
    // Cada thread processará um ponto específico da grade 3D
    // Block Idx = posição do bloco no grid
    // Block Dim = número de threads por bloco
    // Thread Idx = posição da thread dentro do bloco
    int thread_x = blockIdx.x * BLOCK_SIZE_X + threadIdx.x;  // Posição X da thread no grid global
    int thread_y = blockIdx.y * BLOCK_SIZE_Y + threadIdx.y;  // Posição Y da thread no grid global
    int thread_z = blockIdx.z * BLOCK_SIZE_Z + threadIdx.z;  // Posição Z da thread no grid global

    int tile_x = threadIdx.x + RADIUS; // Posição X da thread dentro do bloco
    int tile_y = threadIdx.y + RADIUS; // Posição Y da thread dentro do bloco
    int tile_z = threadIdx.z + RADIUS; // Posição Z da thread dentro do bloco

    // Verifica se a thread está dentro dos limites válidos da grade
    // Exclui as bordas (pontos nas extremidades) pois eles não têm todos os vizinhos necessários
    // Para aplicar o operador de diferenças finitas, precisamos de 6 vizinhos (±X, ±Y, ±Z)
    if (thread_x > 0 && thread_x < grid_points_x-1 &&     // Não está na borda esquerda/direita
        thread_y > 0 && thread_y < grid_points_y-1 &&     // Não está na borda frente/trás  
        thread_z > 0 && thread_z < grid_points_z-1) {     // Não está na borda superior/inferior
        
        
        int current_index = thread_z * grid_points_y * grid_points_x + thread_y * grid_points_x + thread_x;
        
        // Copia pixel central
        if (tile_x < BLOCK_SIZE_X && tile_y < BLOCK_SIZE_Y && tile_z < BLOCK_SIZE_Z)
            tile[tile_z][tile_y][tile_x] = temperature_field_old[current_index];

        // Copia vizinhos para a memória compartilhada
        if (tile_x - 1 >= 0 && tile_x - 1 < BLOCK_SIZE_X && tile_y < BLOCK_SIZE_Y && tile_z < BLOCK_SIZE_Z)
            tile[tile_z][tile_y][tile_x - 1] = temperature_field_old[current_index - 1]; // Vizinho à esquerda
        if (tile_x + 1 < BLOCK_SIZE_X && tile_y < BLOCK_SIZE_Y && tile_z < BLOCK_SIZE_Z)
            tile[tile_z][tile_y][tile_x + 1] = temperature_field_old[current_index + 1]; // Vizinho à direita
        if (tile_y - 1 >= 0 && tile_x < BLOCK_SIZE_X && tile_z < BLOCK_SIZE_Z)
            tile[tile_z][tile_y - 1][tile_x] = temperature_field_old[current_index - grid_points_x]; // Vizinho acima
        if (tile_y + 1 < BLOCK_SIZE_Y && tile_x < BLOCK_SIZE_X && tile_z < BLOCK_SIZE_Z)
            tile[tile_z][tile_y + 1][tile_x] = temperature_field_old[current_index + grid_points_x]; // Vizinho abaixo
        if (tile_z - 1 >= 0 && tile_x < BLOCK_SIZE_X && tile_y < BLOCK_SIZE_Y)
            tile[tile_z - 1][tile_y][tile_x] = temperature_field_old[current_index - grid_points_x * grid_points_y]; // Vizinho atrás
        if (tile_z + 1 < BLOCK_SIZE_Z && tile_x < BLOCK_SIZE_X && tile_y < BLOCK_SIZE_Y)
            tile[tile_z + 1][tile_y][tile_x] = temperature_field_old[current_index + grid_points_x * grid_points_y]; // Vizinho à frente

        __syncthreads();
        
        // Aplica a equação no tile shared
        temperature_field_new[current_index] = tile[tile_z][tile_y][tile_x] + diffusion_alpha * (
            tile[tile_z][tile_y][tile_x - 1] + tile[tile_z][tile_y][tile_x + 1] +   
            tile[tile_z][tile_y - 1][tile_x] + tile[tile_z][tile_y + 1][tile_x] +    
            tile[tile_z - 1][tile_y][tile_x] + tile[tile_z + 1][tile_y][tile_x] -   
            6 * tile[tile_z][tile_y][tile_x]);                                         
    }
}

float calculate_max_deviation_from_one(double* temperature_field, int grid_points_x, int grid_points_y, int grid_points_z) {
    float max_deviation = 0.0f;
    int total_points = grid_points_x * grid_points_y * grid_points_z;
    
    for (int point_index = 0; point_index < total_points; point_index++) {
        float current_deviation = fabs(temperature_field[point_index] - 1.0);
        if (current_deviation > max_deviation) {
            max_deviation = current_deviation;
        }
    }
    return max_deviation;
}

float run_cuda_simulation(int grid_points_x, int grid_points_y, int grid_points_z, int number_of_time_steps, double diffusion_alpha) {
    size_t memory_size = grid_points_x * grid_points_y * grid_points_z * sizeof(double);
    
    // Allocate host memory
    double *host_temperature_initial = (double*)malloc(memory_size);
    double *host_temperature_result = (double*)malloc(memory_size);
    
    if (!host_temperature_initial || !host_temperature_result) {
        printf("Failed to allocate host memory\n");
        return -1.0f;
    }
    
    // Initialize host array: all points to 1.0
    int total_grid_points = grid_points_x * grid_points_y * grid_points_z;
    for (int point_index = 0; point_index < total_grid_points; point_index++) {
        host_temperature_initial[point_index] = 1.0;
    }
    
    // Add perturbation at center
    int center_x_coordinate = grid_points_x / 2;
    int center_y_coordinate = grid_points_y / 2;
    int center_z_coordinate = grid_points_z / 2;
    int center_point_index = center_z_coordinate * grid_points_y * grid_points_x + 
                            center_y_coordinate * grid_points_x + center_x_coordinate;
    host_temperature_initial[center_point_index] += PERTURBATION_MAGNITUDE;
    
    // Allocate device memory
    double *device_temperature_old, *device_temperature_new;
    hipMalloc(&device_temperature_old, memory_size);
    hipMalloc(&device_temperature_new, memory_size);
    
    // Copy initial data to device
    hipMemcpy(device_temperature_old, host_temperature_initial, memory_size, hipMemcpyHostToDevice);
    
    // Define 3D block and grid
    dim3 threads_per_block(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    dim3 blocks_per_grid((grid_points_x + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, 
                        (grid_points_y + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, 
                        (grid_points_z + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
    
    printf("Starting CUDA simulation...\n");
    printf("Grid size: %dx%dx%d\n", grid_points_x, grid_points_y, grid_points_z);
    printf("Block size: %dx%dx%d\n", BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    printf("Grid dimensions: %dx%dx%d\n", blocks_per_grid.x, blocks_per_grid.y, blocks_per_grid.z);
    
    // Create CUDA events for timing
    hipEvent_t simulation_start_event, simulation_stop_event;
    hipEventCreate(&simulation_start_event);
    hipEventCreate(&simulation_stop_event);
    
    // Record start event
    hipEventRecord(simulation_start_event);
    
    // Main simulation loop
    for (int time_step = 0; time_step < number_of_time_steps; time_step++) {
        // Launch kernel
        update_temperature_field<<<blocks_per_grid, threads_per_block>>>(
            device_temperature_new, device_temperature_old, 
            grid_points_x, grid_points_y, grid_points_z, diffusion_alpha);
        
        // Check for kernel launch errors
        hipError_t kernel_error = hipGetLastError();
        if (kernel_error != hipSuccess) {
            printf("CUDA kernel error: %s\n", hipGetErrorString(kernel_error));
            break;
        }
        
        // Swap pointers
        double* temporary_pointer = device_temperature_old;
        device_temperature_old = device_temperature_new;
        device_temperature_new = temporary_pointer;
    }
    
    // Record stop event
    hipEventRecord(simulation_stop_event);
    
    // Wait for all kernels to complete
    hipEventSynchronize(simulation_stop_event);
    
    // Calculate elapsed time using CUDA events
    float elapsed_milliseconds = 0;
    hipEventElapsedTime(&elapsed_milliseconds, simulation_start_event, simulation_stop_event);
    float elapsed_seconds = elapsed_milliseconds / 1000.0f; // Convert to seconds
    
    // Copy result back to host
    hipMemcpy(host_temperature_result, device_temperature_old, memory_size, hipMemcpyDeviceToHost);
    
    // Calculate final deviation
    float final_maximum_deviation = calculate_max_deviation_from_one(host_temperature_result, grid_points_x, grid_points_y, grid_points_z);
    
    printf("Total CUDA simulation time: %f seconds\n", elapsed_seconds);
    printf("Final max deviation from 1.0: %.6f\n", final_maximum_deviation);
    printf("Center value: %.6f\n", host_temperature_result[center_point_index]);
    // Resultado CPU 7.388917

    // Cleanup CUDA events
    hipEventDestroy(simulation_start_event);
    hipEventDestroy(simulation_stop_event);
    
    // Cleanup memory
    hipFree(device_temperature_old);
    hipFree(device_temperature_new);
    free(host_temperature_initial);
    free(host_temperature_result);
    
    return elapsed_seconds;
}
// Adicione esta função CPU ao seu navier.cu
void update_temperature_cpu(double* temp_new, double* temp_old, 
                           int nx, int ny, int nz, double alpha) {
    for (int z = 1; z < nz - 1; z++) {
        for (int y = 1; y < ny - 1; y++) {
            for (int x = 1; x < nx - 1; x++) {
                int idx = z * ny * nx + y * nx + x;
                int idx_xm = idx - 1;
                int idx_xp = idx + 1;
                int idx_ym = idx - nx;
                int idx_yp = idx + nx;
                int idx_zm = idx - nx * ny;
                int idx_zp = idx + nx * ny;
                
                temp_new[idx] = temp_old[idx] + alpha * (
                    temp_old[idx_xm] + temp_old[idx_xp] +
                    temp_old[idx_ym] + temp_old[idx_yp] +
                    temp_old[idx_zm] + temp_old[idx_zp] -
                    6.0 * temp_old[idx]);
            }
        }
    }
}

float run_cpu_simulation(int nx, int ny, int nz, int num_steps, double alpha) {
    size_t size = nx * ny * nz * sizeof(double);
    
    double *cpu_temp_curr = (double*)malloc(size);
    double *cpu_temp_next = (double*)malloc(size);
    
    if (!cpu_temp_curr || !cpu_temp_next) {
        printf("Failed to allocate CPU memory\n");
        return -1.0f;
    }
    
    // Initialize: all points to 1.0
    for (int i = 0; i < nx * ny * nz; i++) {
        cpu_temp_curr[i] = 1.0;
    }
    
    // Add perturbation at center
    int center_idx = (nz/2) * ny * nx + (ny/2) * nx + (nx/2);
    cpu_temp_curr[center_idx] += PERTURBATION_MAGNITUDE;
    
    // Start timing
    struct timeval start, end;
    gettimeofday(&start, NULL);
    
    // Simulation loop
    for (int t = 0; t < num_steps; t++) {
        update_temperature_cpu(cpu_temp_next, cpu_temp_curr, nx, ny, nz, alpha);
        
        // Swap pointers
        double* temp = cpu_temp_curr;
        cpu_temp_curr = cpu_temp_next;
        cpu_temp_next = temp;
    }
    
    // End timing
    gettimeofday(&end, NULL);
    double elapsed = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) * 1e-6;
    
    // Calculate results
    float max_dev = calculate_max_deviation_from_one(cpu_temp_curr, nx, ny, nz);
    
    printf("CPU Results:\n");
    printf("Total CPU simulation time: %f seconds\n", elapsed);
    printf("Final max deviation from 1.0: %.6f\n", max_dev);
    printf("Center value: %.6f\n", cpu_temp_curr[center_idx]);
    
    free(cpu_temp_curr);
    free(cpu_temp_next);
    
    return elapsed;
}
bool compare_results(double* gpu_result, double* cpu_result, int nx, int ny, int nz, double tolerance = 1e-6) {
    int total_points = nx * ny * nz;
    int differences = 0;
    double max_error = 0.0;
    double sum_error = 0.0;
    
    for (int i = 0; i < total_points; i++) {
        double error = fabs(gpu_result[i] - cpu_result[i]);
        if (error > tolerance) {
            differences++;
        }
        if (error > max_error) {
            max_error = error;
        }
        sum_error += error;
    }
    
    double avg_error = sum_error / total_points;
    
    printf("\n=== VALIDATION RESULTS ===\n");
    printf("Total points: %d\n", total_points);
    printf("Points with differences > %.2e: %d (%.2f%%)\n", 
           tolerance, differences, 100.0 * differences / total_points);
    printf("Maximum error: %.2e\n", max_error);
    printf("Average error: %.2e\n", avg_error);
    printf("Tolerance: %.2e\n", tolerance);
    
    bool passed = (differences == 0) || (max_error < tolerance);
    printf("Validation: %s\n", passed ? "PASSED" : "FAILED");
    
    return passed;
}
int main() {
    // Simulation parameters
    int nx = GRID_SIZE_X;
    int ny = GRID_SIZE_Y;
    int nz = GRID_SIZE_Z;
    int nt = NUM_TIME_STEPS;
    double alpha = DIFFUSION_COEFFICIENT * TIME_STEP_INCREMENT / 
                   (CELL_SPACING_X * CELL_SPACING_X);
    
    printf("## Comparação CPU vs GPU - Difusão 3D ##\n");
    printf("Grid: %dx%dx%d\n", nx, ny, nz);
    printf("Time steps: %d\n", nt);
    printf("Alpha: %f\n", alpha);
    printf("========================================\n");
    
    // Check CUDA device
    int device_count;
    hipGetDeviceCount(&device_count);
    if (device_count == 0) {
        printf("No CUDA devices found! Running CPU only.\n");
        run_cpu_simulation(nx, ny, nz, nt, alpha);
        return 1;
    }
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Using GPU: %s\n", prop.name);
    printf("========================================\n");
    
    // Run CPU simulation
    printf("\n>>> Running CPU simulation...\n");
    float cpu_time = run_cpu_simulation(nx, ny, nz, nt, alpha);
    
    printf("\n>>> Running GPU simulation...\n");
    float gpu_time = run_cuda_simulation(nx, ny, nz, nt, alpha);
    
    // Performance comparison
    if (cpu_time > 0 && gpu_time > 0) {
        printf("\n=== PERFORMANCE COMPARISON ===\n");
        printf("CPU time: %.6f seconds\n", cpu_time);
        printf("GPU time: %.6f seconds\n", gpu_time);
        printf("Speedup: %.2fx\n", cpu_time / gpu_time);
    }
    
    return 0;
}