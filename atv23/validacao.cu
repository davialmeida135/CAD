#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <omp.h>
// --- Configuration Constants ---
#define GRID_SIZE_X 512        // Number of grid points in X dimension
#define GRID_SIZE_Y 512        // Number of grid points in Y dimension
#define GRID_SIZE_Z 512        // Number of grid points in Z dimension
#define NUM_TIME_STEPS 10      // Total number of simulation time steps
#define CELL_SPACING_X 1.0f     // Spatial step size (delta X)
#define CELL_SPACING_Y 1.0f     // Spatial step size (delta Y)
#define CELL_SPACING_Z 1.0f     // Spatial step size (delta Z)
#define TIME_STEP_INCREMENT 0.1f // Time step size (delta T)
#define DIFFUSION_COEFFICIENT 0.5f // Diffusion coefficient (Nu)
#define PERTURBATION_MAGNITUDE 100.0f // Magnitude of perturbation
#define RADIUS 1
#define BLOCK_SIZE_X 8 // Number of threads in X dimension per block
#define BLOCK_SIZE_Y 8 // Number of threads in Y dimension per block
#define BLOCK_SIZE_Z 8 // Number of threads in Z dimension per block

// Macro to convert 3D grid coordinates to a 1D array index
#define MAP_3D_TO_1D_INDEX(x_coord, y_coord, z_coord) ((z_coord) * GRID_SIZE_Y * GRID_SIZE_X + (y_coord) * GRID_SIZE_X + (x_coord))

__global__ void update_temperature_field(double* temperature_field_new, double* temperature_field_old, 
    int grid_points_x, int grid_points_y, int grid_points_z, double diffusion_alpha) {

    __shared__ double tile[BLOCK_SIZE_Z + 2*RADIUS][BLOCK_SIZE_Y + 2*RADIUS][BLOCK_SIZE_X + 2*RADIUS];
    
    // Coordenadas da thread (e do ponto no espaço)
    int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_y = blockIdx.y * blockDim.y + threadIdx.y;
    int thread_z = blockIdx.z * blockDim.z + threadIdx.z;
    
    // Posição da thread dentro do bloco
    int local_x = threadIdx.x;
    int local_y = threadIdx.y;
    int local_z = threadIdx.z;
    
    // Posição do tile (thread+raio)
    int tile_x = local_x + RADIUS;
    int tile_y = local_y + RADIUS;
    int tile_z = local_z + RADIUS;
    
    // Carregar o ponto da thread
    if (thread_x < grid_points_x && thread_y < grid_points_y && thread_z < grid_points_z) {
        int global_idx = thread_z * grid_points_y * grid_points_x + thread_y * grid_points_x + thread_x;
        tile[tile_z][tile_y][tile_x] = temperature_field_old[global_idx];
    } else {
        tile[tile_z][tile_y][tile_x] = 1.0;
    }
    
    // Carregar os halos
    // Eixo X
    // Apenas a primeira thread de cada dimensão carregam os halos, não a primeira e a ultima
    if (local_x < RADIUS) {
        // Da esquerda (posicao - raio)
        int halo_x = thread_x - RADIUS;
        if (halo_x >= 0 && thread_y < grid_points_y && thread_z < grid_points_z) {
            int halo_idx = thread_z * grid_points_y * grid_points_x + thread_y * grid_points_x + halo_x;
            tile[tile_z][tile_y][local_x] = temperature_field_old[halo_idx];
        } else {
            tile[tile_z][tile_y][local_x] = 1.0;
        }
        
        // Da dureita (posicao + tamanho)
        int right_x = thread_x + blockDim.x;
        if (right_x < grid_points_x && thread_y < grid_points_y && thread_z < grid_points_z) {
            int right_idx = thread_z * grid_points_y * grid_points_x + thread_y * grid_points_x + right_x;
            tile[tile_z][tile_y][local_x + blockDim.x + RADIUS] = temperature_field_old[right_idx];
        } else {
            tile[tile_z][tile_y][local_x + blockDim.x + RADIUS] = 1.0;
        }
    }
    // Eixo Y
    if (local_y < RADIUS) {
        // De frente (posicao - raio)
        int halo_y = thread_y - RADIUS;
        if (halo_y >= 0 && thread_x < grid_points_x && thread_z < grid_points_z) {
            int halo_idx = thread_z * grid_points_y * grid_points_x + halo_y * grid_points_x + thread_x;
            tile[tile_z][local_y][tile_x] = temperature_field_old[halo_idx];
        } else {
            tile[tile_z][local_y][tile_x] = 1.0;
        }
        
        // De trás (posicao + tamanho)
        int back_y = thread_y + blockDim.y;
        if (back_y < grid_points_y && thread_x < grid_points_x && thread_z < grid_points_z) {
            int back_idx = thread_z * grid_points_y * grid_points_x + back_y * grid_points_x + thread_x;
            tile[tile_z][local_y + blockDim.y + RADIUS][tile_x] = temperature_field_old[back_idx];
        } else {
            tile[tile_z][local_y + blockDim.y + RADIUS][tile_x] = 1.0;
        }
    }
    // Eixo Z
    if (local_z < RADIUS) {
        // De cima (posicao - raio)
        int halo_z = thread_z - RADIUS;
        if (halo_z >= 0 && thread_x < grid_points_x && thread_y < grid_points_y) {
            int halo_idx = halo_z * grid_points_y * grid_points_x + thread_y * grid_points_x + thread_x;
            tile[local_z][tile_y][tile_x] = temperature_field_old[halo_idx];
        } else {
            tile[local_z][tile_y][tile_x] = 1.0;
        }
        
        // De baixo (posicao - raio)
        int top_z = thread_z + blockDim.z;
        if (top_z < grid_points_z && thread_x < grid_points_x && thread_y < grid_points_y) {
            int top_idx = top_z * grid_points_y * grid_points_x + thread_y * grid_points_x + thread_x;
            tile[local_z + blockDim.z + RADIUS][tile_y][tile_x] = temperature_field_old[top_idx];
        } else {
            tile[local_z + blockDim.z + RADIUS][tile_y][tile_x] = 1.0;
        }
    }
    __syncthreads();
    // Calcular somente para pontos internos (nao fazer para os halos)
    if (thread_x > 0 && thread_x < grid_points_x-1 &&
        thread_y > 0 && thread_y < grid_points_y-1 &&
        thread_z > 0 && thread_z < grid_points_z-1) {
        
        int global_idx = thread_z * grid_points_y * grid_points_x + thread_y * grid_points_x + thread_x;
        
        double center = tile[tile_z][tile_y][tile_x];
        double neighbors = tile[tile_z][tile_y][tile_x-1] + tile[tile_z][tile_y][tile_x+1] +  // direita - esquerda
                          tile[tile_z][tile_y-1][tile_x] + tile[tile_z][tile_y+1][tile_x] +  // cima - baixo  
                          tile[tile_z-1][tile_y][tile_x] + tile[tile_z+1][tile_y][tile_x];   // frente - trás
        
        temperature_field_new[global_idx] = center + diffusion_alpha * (neighbors - 6.0 * center);
    }
}

float calculate_max_deviation_from_one(double* temperature_field, int grid_points_x, int grid_points_y, int grid_points_z) {
    float max_deviation = 0.0f;
    int total_points = grid_points_x * grid_points_y * grid_points_z;
    
    for (int point_index = 0; point_index < total_points; point_index++) {
        float current_deviation = fabs(temperature_field[point_index] - 1.0);
        if (current_deviation > max_deviation) {
            max_deviation = current_deviation;
        }
    }
    return max_deviation;
}

float run_cuda_simulation(int grid_points_x, int grid_points_y, int grid_points_z, int number_of_time_steps, double diffusion_alpha) {
    size_t memory_size = grid_points_x * grid_points_y * grid_points_z * sizeof(double);
    
    // Allocate host memory
    double *host_temperature_initial = (double*)malloc(memory_size);
    double *host_temperature_result = (double*)malloc(memory_size);
    
    if (!host_temperature_initial || !host_temperature_result) {
        printf("Failed to allocate host memory\n");
        return -1.0f;
    }
    
    // Initialize host array: all points to 1.0
    int total_grid_points = grid_points_x * grid_points_y * grid_points_z;
    for (int point_index = 0; point_index < total_grid_points; point_index++) {
        host_temperature_initial[point_index] = 1.0;
    }
    
    // Add perturbation at center
    int center_x_coordinate = grid_points_x / 2;
    int center_y_coordinate = grid_points_y / 2;
    int center_z_coordinate = grid_points_z / 2;
    int center_point_index = center_z_coordinate * grid_points_y * grid_points_x + 
                            center_y_coordinate * grid_points_x + center_x_coordinate;
    host_temperature_initial[center_point_index] += PERTURBATION_MAGNITUDE;
    
    // Allocate device memory
    double *device_temperature_old, *device_temperature_new;
    hipMalloc(&device_temperature_old, memory_size);
    hipMalloc(&device_temperature_new, memory_size);
    
    // Copy initial data to device
    hipMemcpy(device_temperature_old, host_temperature_initial, memory_size, hipMemcpyHostToDevice);
    
    // Define 3D block and grid
    dim3 threads_per_block(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    dim3 blocks_per_grid((grid_points_x + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, 
                        (grid_points_y + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, 
                        (grid_points_z + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
    
    printf("Starting CUDA simulation...\n");
    printf("Grid size: %dx%dx%d\n", grid_points_x, grid_points_y, grid_points_z);
    printf("Block size: %dx%dx%d\n", BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    printf("Grid dimensions: %dx%dx%d\n", blocks_per_grid.x, blocks_per_grid.y, blocks_per_grid.z);
    
    // Create CUDA events for timing
    hipEvent_t simulation_start_event, simulation_stop_event;
    hipEventCreate(&simulation_start_event);
    hipEventCreate(&simulation_stop_event);
    
    // Record start event
    hipEventRecord(simulation_start_event);
    
    // Main simulation loop
    for (int time_step = 0; time_step < number_of_time_steps; time_step++) {
        // Launch kernel
        update_temperature_field<<<blocks_per_grid, threads_per_block>>>(
            device_temperature_new, device_temperature_old, 
            grid_points_x, grid_points_y, grid_points_z, diffusion_alpha);
        
        // Check for kernel launch errors
        hipError_t kernel_error = hipGetLastError();
        if (kernel_error != hipSuccess) {
            printf("CUDA kernel error: %s\n", hipGetErrorString(kernel_error));
            break;
        }
        
        // Swap pointers
        double* temporary_pointer = device_temperature_old;
        device_temperature_old = device_temperature_new;
        device_temperature_new = temporary_pointer;
    }
    
    // Record stop event
    hipEventRecord(simulation_stop_event);
    
    // Wait for all kernels to complete
    hipEventSynchronize(simulation_stop_event);
    
    // Calculate elapsed time using CUDA events
    float elapsed_milliseconds = 0;
    hipEventElapsedTime(&elapsed_milliseconds, simulation_start_event, simulation_stop_event);
    float elapsed_seconds = elapsed_milliseconds / 1000.0f; // Convert to seconds
    
    // Copy result back to host
    hipMemcpy(host_temperature_result, device_temperature_old, memory_size, hipMemcpyDeviceToHost);
    
    // Calculate final deviation
    float final_maximum_deviation = calculate_max_deviation_from_one(host_temperature_result, grid_points_x, grid_points_y, grid_points_z);
    
    printf("Total CUDA simulation time: %f seconds\n", elapsed_seconds);
    printf("Final max deviation from 1.0: %.6f\n", final_maximum_deviation);
    printf("Center value: %.6f\n", host_temperature_result[center_point_index]);
    // Resultado CPU 7.388917

    // Cleanup CUDA events
    hipEventDestroy(simulation_start_event);
    hipEventDestroy(simulation_stop_event);
    
    // Cleanup memory
    hipFree(device_temperature_old);
    hipFree(device_temperature_new);
    free(host_temperature_initial);
    free(host_temperature_result);
    
    return elapsed_seconds;
}
// Adicione esta função CPU ao seu navier.cu
void update_temperature_cpu(double* temp_new, double* temp_old, 
                           int nx, int ny, int nz, double alpha) {
    #pragma omp parallel for num_threads(16)
    for (int z = 1; z < nz - 1; z++) {
        for (int y = 1; y < ny - 1; y++) {
            for (int x = 1; x < nx - 1; x++) {
                int idx = z * ny * nx + y * nx + x;
                int idx_xm = idx - 1;
                int idx_xp = idx + 1;
                int idx_ym = idx - nx;
                int idx_yp = idx + nx;
                int idx_zm = idx - nx * ny;
                int idx_zp = idx + nx * ny;
                
                temp_new[idx] = temp_old[idx] + alpha * (
                    temp_old[idx_xm] + temp_old[idx_xp] +
                    temp_old[idx_ym] + temp_old[idx_yp] +
                    temp_old[idx_zm] + temp_old[idx_zp] -
                    6.0 * temp_old[idx]);
            }
        }
    }
}

float run_cpu_simulation(int nx, int ny, int nz, int num_steps, double alpha) {
    size_t size = nx * ny * nz * sizeof(double);
    
    double *cpu_temp_curr = (double*)malloc(size);
    double *cpu_temp_next = (double*)malloc(size);
    
    if (!cpu_temp_curr || !cpu_temp_next) {
        printf("Failed to allocate CPU memory\n");
        return -1.0f;
    }
    
    // Initialize: all points to 1.0
    for (int i = 0; i < nx * ny * nz; i++) {
        cpu_temp_curr[i] = 1.0;
    }
    
    // Add perturbation at center
    int center_idx = (nz/2) * ny * nx + (ny/2) * nx + (nx/2);
    cpu_temp_curr[center_idx] += PERTURBATION_MAGNITUDE;
    
    // Start timing
    struct timeval start, end;
    gettimeofday(&start, NULL);
    
    // Simulation loop
    for (int t = 0; t < num_steps; t++) {
        update_temperature_cpu(cpu_temp_next, cpu_temp_curr, nx, ny, nz, alpha);
        
        // Swap pointers
        double* temp = cpu_temp_curr;
        cpu_temp_curr = cpu_temp_next;
        cpu_temp_next = temp;
    }
    
    // End timing
    gettimeofday(&end, NULL);
    double elapsed = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) * 1e-6;
    
    // Calculate results
    float max_dev = calculate_max_deviation_from_one(cpu_temp_curr, nx, ny, nz);
    
    printf("CPU Results:\n");
    printf("Total CPU simulation time: %f seconds\n", elapsed);
    printf("Final max deviation from 1.0: %.6f\n", max_dev);
    printf("Center value: %.6f\n", cpu_temp_curr[center_idx]);
    
    free(cpu_temp_curr);
    free(cpu_temp_next);
    
    return elapsed;
}
bool compare_results(double* gpu_result, double* cpu_result, int nx, int ny, int nz, double tolerance = 1e-6) {
    int total_points = nx * ny * nz;
    int differences = 0;
    double max_error = 0.0;
    double sum_error = 0.0;
    
    for (int i = 0; i < total_points; i++) {
        double error = fabs(gpu_result[i] - cpu_result[i]);
        if (error > tolerance) {
            differences++;
        }
        if (error > max_error) {
            max_error = error;
        }
        sum_error += error;
    }
    
    double avg_error = sum_error / total_points;
    
    printf("\n=== VALIDATION RESULTS ===\n");
    printf("Total points: %d\n", total_points);
    printf("Points with differences > %.2e: %d (%.2f%%)\n", 
           tolerance, differences, 100.0 * differences / total_points);
    printf("Maximum error: %.2e\n", max_error);
    printf("Average error: %.2e\n", avg_error);
    printf("Tolerance: %.2e\n", tolerance);
    
    bool passed = (differences == 0) || (max_error < tolerance);
    printf("Validation: %s\n", passed ? "PASSED" : "FAILED");
    
    return passed;
}


// Adicione estas funções antes do main()

// Estrutura para armazenar resultados da simulação
typedef struct {
    double* temperature_field;
    float execution_time;
    float max_deviation;
    double center_value;
} SimulationResult;

// Função para executar simulação CPU e retornar resultados
SimulationResult run_cpu_simulation_with_results(int nx, int ny, int nz, int num_steps, double alpha) {
    SimulationResult result = {0};
    size_t size = nx * ny * nz * sizeof(double);
    
    result.temperature_field = (double*)malloc(size);
    double *cpu_temp_next = (double*)malloc(size);
    
    if (!result.temperature_field || !cpu_temp_next) {
        printf("Failed to allocate CPU memory\n");
        result.execution_time = -1.0f;
        return result;
    }
    
    // Initialize: all points to 1.0
    for (int i = 0; i < nx * ny * nz; i++) {
        result.temperature_field[i] = 1.0;
    }
    
    // Add perturbation at center
    int center_idx = (nz/2) * ny * nx + (ny/2) * nx + (nx/2);
    result.temperature_field[center_idx] += PERTURBATION_MAGNITUDE;
    
    // Start timing
    struct timeval start, end;
    gettimeofday(&start, NULL);
    
    // Simulation loop
    for (int t = 0; t < num_steps; t++) {
        update_temperature_cpu(cpu_temp_next, result.temperature_field, nx, ny, nz, alpha);
        
        // Swap pointers
        double* temp = result.temperature_field;
        result.temperature_field = cpu_temp_next;
        cpu_temp_next = temp;
    }
    
    // End timing
    gettimeofday(&end, NULL);
    result.execution_time = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) * 1e-6;
    
    // Calculate results
    result.max_deviation = calculate_max_deviation_from_one(result.temperature_field, nx, ny, nz);
    result.center_value = result.temperature_field[center_idx];
    
    printf("CPU Results:\n");
    printf("Total CPU simulation time: %f seconds\n", result.execution_time);
    printf("Final max deviation from 1.0: %.6f\n", result.max_deviation);
    printf("Center value: %.6f\n", result.center_value);
    
    free(cpu_temp_next);
    return result;
}

// Função para executar simulação GPU e retornar resultados
SimulationResult run_cuda_simulation_with_results(int grid_points_x, int grid_points_y, int grid_points_z, int number_of_time_steps, double diffusion_alpha) {
    SimulationResult result = {0};
    size_t memory_size = grid_points_x * grid_points_y * grid_points_z * sizeof(double);
    
    // Allocate host memory
    double *host_temperature_initial = (double*)malloc(memory_size);
    result.temperature_field = (double*)malloc(memory_size);
    
    if (!host_temperature_initial || !result.temperature_field) {
        printf("Failed to allocate host memory\n");
        result.execution_time = -1.0f;
        return result;
    }
    
    // Initialize host array: all points to 1.0
    int total_grid_points = grid_points_x * grid_points_y * grid_points_z;
    for (int point_index = 0; point_index < total_grid_points; point_index++) {
        host_temperature_initial[point_index] = 1.0;
    }
    
    // Add perturbation at center
    int center_x_coordinate = grid_points_x / 2;
    int center_y_coordinate = grid_points_y / 2;
    int center_z_coordinate = grid_points_z / 2;
    int center_point_index = center_z_coordinate * grid_points_y * grid_points_x + 
                            center_y_coordinate * grid_points_x + center_x_coordinate;
    host_temperature_initial[center_point_index] += PERTURBATION_MAGNITUDE;
    
    // Allocate device memory
    double *device_temperature_old, *device_temperature_new;
    hipMalloc(&device_temperature_old, memory_size);
    hipMalloc(&device_temperature_new, memory_size);
    
    // Copy initial data to device
    hipMemcpy(device_temperature_old, host_temperature_initial, memory_size, hipMemcpyHostToDevice);
    
    // Define 3D block and grid
    dim3 threads_per_block(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    dim3 blocks_per_grid((grid_points_x + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, 
                        (grid_points_y + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, 
                        (grid_points_z + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
    
    printf("Starting CUDA simulation...\n");
    printf("Grid size: %dx%dx%d\n", grid_points_x, grid_points_y, grid_points_z);
    printf("Block size: %dx%dx%d\n", BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    printf("Grid dimensions: %dx%dx%d\n", blocks_per_grid.x, blocks_per_grid.y, blocks_per_grid.z);
    
    // Create CUDA events for timing
    hipEvent_t simulation_start_event, simulation_stop_event;
    hipEventCreate(&simulation_start_event);
    hipEventCreate(&simulation_stop_event);
    
    // Record start event
    hipEventRecord(simulation_start_event);
    
    // Main simulation loop
    for (int time_step = 0; time_step < number_of_time_steps; time_step++) {
        // Launch kernel
        update_temperature_field<<<blocks_per_grid, threads_per_block>>>(
            device_temperature_new, device_temperature_old, 
            grid_points_x, grid_points_y, grid_points_z, diffusion_alpha);
        
        // Check for kernel launch errors
        hipError_t kernel_error = hipGetLastError();
        if (kernel_error != hipSuccess) {
            printf("CUDA kernel error: %s\n", hipGetErrorString(kernel_error));
            break;
        }
        
        // Swap pointers
        double* temporary_pointer = device_temperature_old;
        device_temperature_old = device_temperature_new;
        device_temperature_new = temporary_pointer;
    }
    
    // Record stop event
    hipEventRecord(simulation_stop_event);
    
    // Wait for all kernels to complete
    hipEventSynchronize(simulation_stop_event);
    
    // Calculate elapsed time using CUDA events
    float elapsed_milliseconds = 0;
    hipEventElapsedTime(&elapsed_milliseconds, simulation_start_event, simulation_stop_event);
    result.execution_time = elapsed_milliseconds / 1000.0f; // Convert to seconds
    
    // Copy result back to host
    hipMemcpy(result.temperature_field, device_temperature_old, memory_size, hipMemcpyDeviceToHost);
    
    // Calculate final deviation
    result.max_deviation = calculate_max_deviation_from_one(result.temperature_field, grid_points_x, grid_points_y, grid_points_z);
    result.center_value = result.temperature_field[center_point_index];
    
    printf("Total CUDA simulation time: %f seconds\n", result.execution_time);
    printf("Final max deviation from 1.0: %.6f\n", result.max_deviation);
    printf("Center value: %.6f\n", result.center_value);
    
    // Cleanup CUDA events
    hipEventDestroy(simulation_start_event);
    hipEventDestroy(simulation_stop_event);
    
    // Cleanup memory
    hipFree(device_temperature_old);
    hipFree(device_temperature_new);
    free(host_temperature_initial);
    
    return result;
}

// Função completa de validação com múltiplas métricas
typedef struct {
    bool validation_passed;
    int total_points;
    int points_with_differences;
    double max_absolute_error;
    double mean_absolute_error;
    double root_mean_square_error;
    double relative_error_percent;
    double correlation_coefficient;
    double tolerance_used;
} ValidationMetrics;

ValidationMetrics validate_simulation_results(SimulationResult cpu_result, SimulationResult gpu_result, 
                                             int nx, int ny, int nz, double tolerance = 1e-6) {
    ValidationMetrics metrics = {0};
    int total_points = nx * ny * nz;
    metrics.total_points = total_points;
    metrics.tolerance_used = tolerance;
    
    double sum_absolute_error = 0.0;
    double sum_squared_error = 0.0;
    double sum_cpu = 0.0;
    double sum_gpu = 0.0;
    double sum_cpu_squared = 0.0;
    double sum_gpu_squared = 0.0;
    double sum_cpu_gpu = 0.0;
    
    // Calculate all error metrics in one pass
    for (int i = 0; i < total_points; i++) {
        double cpu_val = cpu_result.temperature_field[i];
        double gpu_val = gpu_result.temperature_field[i];
        
        double absolute_error = fabs(gpu_val - cpu_val);
        
        // Count points exceeding tolerance
        if (absolute_error > tolerance) {
            metrics.points_with_differences++;
        }
        
        // Track maximum error
        if (absolute_error > metrics.max_absolute_error) {
            metrics.max_absolute_error = absolute_error;
        }
        
        // Accumulate for various metrics
        sum_absolute_error += absolute_error;
        sum_squared_error += absolute_error * absolute_error;
        
        // For correlation coefficient
        sum_cpu += cpu_val;
        sum_gpu += gpu_val;
        sum_cpu_squared += cpu_val * cpu_val;
        sum_gpu_squared += gpu_val * gpu_val;
        sum_cpu_gpu += cpu_val * gpu_val;
    }
    
    // Calculate derived metrics
    metrics.mean_absolute_error = sum_absolute_error / total_points;
    metrics.root_mean_square_error = sqrt(sum_squared_error / total_points);
    
    // Relative error as percentage
    double mean_cpu = sum_cpu / total_points;
    if (mean_cpu != 0.0) {
        metrics.relative_error_percent = (metrics.mean_absolute_error / fabs(mean_cpu)) * 100.0;
    }
    
    // Correlation coefficient (Pearson)
    double mean_cpu_calc = sum_cpu / total_points;
    double mean_gpu_calc = sum_gpu / total_points;
    
    double numerator = sum_cpu_gpu - total_points * mean_cpu_calc * mean_gpu_calc;
    double denominator = sqrt((sum_cpu_squared - total_points * mean_cpu_calc * mean_cpu_calc) * 
                             (sum_gpu_squared - total_points * mean_gpu_calc * mean_gpu_calc));
    
    if (denominator != 0.0) {
        metrics.correlation_coefficient = numerator / denominator;
    } else {
        metrics.correlation_coefficient = 1.0; // Perfect correlation if both are constant
    }
    
    // Determine if validation passed
    metrics.validation_passed = (metrics.max_absolute_error < tolerance) && 
                               (metrics.correlation_coefficient > 0.99999); // Very high correlation required
    
    return metrics;
}

void print_validation_report(ValidationMetrics metrics, SimulationResult cpu_result, SimulationResult gpu_result) {
    printf("\n=== VALIDATION REPORT ===\n");
    printf("========================================\n");
    
    printf("Dataset Statistics:\n");
    printf("  Total points analyzed: %d\n", metrics.total_points);
    printf("  Tolerance threshold: %.2e\n", metrics.tolerance_used);
    printf("\n");
    
    printf("Error Analysis:\n");
    printf("  Maximum absolute error: %.2e\n", metrics.max_absolute_error);
    printf("  Mean absolute error: %.2e\n", metrics.mean_absolute_error);
    printf("  Root mean square error: %.2e\n", metrics.root_mean_square_error);
    printf("  Relative error: %.6f%%\n", metrics.relative_error_percent);
    printf("  Points exceeding tolerance: %d (%.4f%%)\n", 
           metrics.points_with_differences, 
           100.0 * metrics.points_with_differences / metrics.total_points);
    printf("\n");
    
    printf("Statistical Correlation:\n");
    printf("  Pearson correlation coefficient: %.10f\n", metrics.correlation_coefficient);
    printf("\n");
    
    printf("Key Value Comparison:\n");
    printf("  CPU center value: %.10f\n", cpu_result.center_value);
    printf("  GPU center value: %.10f\n", gpu_result.center_value);
    printf("  Center value difference: %.2e\n", fabs(cpu_result.center_value - gpu_result.center_value));
    printf("  CPU max deviation: %.10f\n", cpu_result.max_deviation);
    printf("  GPU max deviation: %.10f\n", gpu_result.max_deviation);
    printf("  Max deviation difference: %.2e\n", fabs(cpu_result.max_deviation - gpu_result.max_deviation));
    printf("\n");
    
    printf("Performance Comparison:\n");
    printf("  CPU execution time: %.6f seconds\n", cpu_result.execution_time);
    printf("  GPU execution time: %.6f seconds\n", gpu_result.execution_time);
    if (cpu_result.execution_time > 0 && gpu_result.execution_time > 0) {
        printf("  Speedup (CPU/GPU): %.2fx\n", cpu_result.execution_time / gpu_result.execution_time);
    }
    printf("\n");
    
    
    if (!metrics.validation_passed) {
        printf("\nPossible causes of validation failure:\n");
        if (metrics.max_absolute_error >= metrics.tolerance_used) {
            printf("  - Numerical errors exceed tolerance\n");
        }
        if (metrics.correlation_coefficient < 0.99999) {
            printf("  - Low correlation between CPU and GPU results\n");
        }
        printf("  - Consider adjusting tolerance or checking implementation\n");
    }
    
    printf("========================================\n");
}

// Função para salvar resultados detalhados (opcional)
void save_detailed_comparison(SimulationResult cpu_result, SimulationResult gpu_result, 
                             ValidationMetrics metrics, int nx, int ny, int nz) {
    FILE *file = fopen("validation_report.txt", "w");
    if (!file) {
        printf("Warning: Could not create detailed report file\n");
        return;
    }
    
    fprintf(file, "CUDA Validation Report\n");
    fprintf(file, "======================\n\n");
    fprintf(file, "Grid dimensions: %dx%dx%d\n", nx, ny, nz);
    fprintf(file, "Total points: %d\n", metrics.total_points);
    
    fprintf(file, "Error Metrics:\n");
    fprintf(file, "Max absolute error: %.2e\n", metrics.max_absolute_error);
    fprintf(file, "Mean absolute error: %.2e\n", metrics.mean_absolute_error);
    fprintf(file, "RMSE: %.2e\n", metrics.root_mean_square_error);
    fprintf(file, "Correlation: %.10f\n", metrics.correlation_coefficient);
    
    fprintf(file, "Performance:\n");
    fprintf(file, "CPU time: %.6f s\n", cpu_result.execution_time);
    fprintf(file, "GPU time: %.6f s\n", gpu_result.execution_time);
    if (cpu_result.execution_time > 0 && gpu_result.execution_time > 0) {
        fprintf(file, "Speedup: %.2fx\n", cpu_result.execution_time / gpu_result.execution_time);
    }
    
    fclose(file);
    printf("Detailed report saved to: validation_report.txt\n");
}

// Modificar o main() para usar as novas funções
int main() {
    // Simulation parameters
    int nx = GRID_SIZE_X;
    int ny = GRID_SIZE_Y;
    int nz = GRID_SIZE_Z;
    int nt = NUM_TIME_STEPS;
    double alpha = DIFFUSION_COEFFICIENT * TIME_STEP_INCREMENT / 
                   (CELL_SPACING_X * CELL_SPACING_X);
    
    printf("## Validação CPU vs GPU - Difusão 3D ##\n");
    printf("Grid: %dx%dx%d\n", nx, ny, nz);
    printf("Time steps: %d\n", nt);
    printf("Alpha: %f\n", alpha);
    printf("========================================\n");
    
    // Check CUDA device
    int device_count;
    hipGetDeviceCount(&device_count);
    if (device_count == 0) {
        printf("No CUDA devices found! Running CPU only.\n");
        SimulationResult cpu_only = run_cpu_simulation_with_results(nx, ny, nz, nt, alpha);
        free(cpu_only.temperature_field);
        return 1;
    }
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Using GPU: %s\n", prop.name);
    printf("========================================\n");
    
    // Run both simulations
    printf("\n>>> Running CPU simulation...\n");
    SimulationResult cpu_result = run_cpu_simulation_with_results(nx, ny, nz, nt, alpha);
    
    printf("\n>>> Running GPU simulation...\n");
    SimulationResult gpu_result = run_cuda_simulation_with_results(nx, ny, nz, nt, alpha);
    
    // Validate results
    if (cpu_result.execution_time > 0 && gpu_result.execution_time > 0) {
        ValidationMetrics validation = validate_simulation_results(cpu_result, gpu_result, nx, ny, nz, 1e-6);
        print_validation_report(validation, cpu_result, gpu_result);
        save_detailed_comparison(cpu_result, gpu_result, validation, nx, ny, nz);
    }
    
    // Cleanup
    free(cpu_result.temperature_field);
    free(gpu_result.temperature_field);
    
    return 0;
}